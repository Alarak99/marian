#include "hip/hip_runtime.h"
#include "matrix.h"

namespace amunmt {
namespace GPU {
namespace mblas {

__global__ void gSum(const float *data, size_t count, float &ret)
{
  ret = 0.0f;
  for (size_t i = 0; i < count; ++i) {
    ret += data[i];
  }
}

float Sum(const float *data, size_t count)
{
  float ret;
  float *d_ret;
  HANDLE_ERROR( hipMalloc((void**)&d_ret, sizeof(float)) );

  HANDLE_ERROR( hipStreamSynchronize(CudaStreamHandler::GetStream()));

  gSum<<<1,1>>>(data, count, *d_ret);
  HANDLE_ERROR( hipMemcpy(&ret, d_ret, sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR(hipFree(d_ret));

  HANDLE_ERROR( hipStreamSynchronize(CudaStreamHandler::GetStream()));

  return ret;
}

}
}
}
