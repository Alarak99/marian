#include "hip/hip_runtime.h"
#include "gpu/mblas/nth_element.h"
#include <iostream>


namespace GPU {

static void HandleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    std::cerr << "ERROR: " << hipGetErrorString(err) << " in " << file << " at line " << line << std::endl;
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void gMaxElement(float* d_out, int* d_ind, float* d_in, int in_size) {
  extern __shared__ float sdata[];
  __shared__ int indices[512];


  int tid = threadIdx.x;
  int i = blockIdx.x * (blockDim.x * 2) + tid;

  sdata[tid] = -3.40282e+38f;

  if (i >= in_size) return;

  if (i + blockDim.x < in_size) {
    float a = d_in[i];
    float b = d_in[i+blockDim.x];
    if (a > b) {
      sdata[tid] = a;
      indices[tid] = i;
    } else {
      sdata[tid] = b;
      indices[tid] = i + blockDim.x;
    }
  } else {
    sdata[tid] = d_in[i];
    indices[tid] = i;
  }

  while (i + 2 * gridDim.x * blockDim.x < in_size) {
    i += 2 * gridDim.x * blockDim.x;

    float a = d_in[i];
    if (a > sdata[tid]) {
      sdata[tid] = a;
      indices[tid] = i;
    }

    if (i + blockDim.x < in_size) {
      float b = d_in[i + blockDim.x];
      if (b > sdata[tid]) {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }
  }
  i = blockIdx.x * (blockDim.x * 2) + tid;

  __syncthreads();

  for (int s = (blockDim.x >> 1); s > 32; s >>= 1) {
    if (tid < s && tid + s < in_size) {
      if (sdata[tid + s] > sdata[tid]) {
        sdata[tid] = sdata[tid + s];
        indices[tid] = indices[tid + s];
      }
    }
    __syncthreads();
  }

  if (tid < 32 && tid + 32 < in_size) {
    if (sdata[tid + 32] > sdata[tid]) {
      sdata[tid] = sdata[tid + 32];
      indices[tid] = indices[tid + 32];
    }
  }

  if (tid < 16 && tid + 16 < in_size) {
    if (sdata[tid + 16] > sdata[tid]) {
      sdata[tid] = sdata[tid + 16];
      indices[tid] = indices[tid + 16];
    }
  }

  if (tid < 8 && tid + 8 < in_size) {
    if (sdata[tid + 8] > sdata[tid]) {
      sdata[tid] = sdata[tid + 8];
      indices[tid] = indices[tid + 8];
    }
  }

  if (tid < 4 && tid + 4 < in_size) {
    if (sdata[tid + 4] > sdata[tid]) {
      sdata[tid] = sdata[tid + 4];
      indices[tid] = indices[tid + 4];
    }
  }

  if (tid < 2 && tid + 2 < in_size) {
    if (sdata[tid + 2] > sdata[tid]) {
      sdata[tid] = sdata[tid + 2];
      indices[tid] = indices[tid + 2];
    }
  }

  if (tid < 1 && tid + 1 < in_size) {
    if (sdata[tid + 1] > sdata[tid]) {
      sdata[tid] = sdata[tid + 1];
      indices[tid] = indices[tid + 1];
    }
  }

  if (tid == 0) {
    d_out[blockIdx.x] = sdata[0];
    d_ind[blockIdx.x] = indices[0];
  }
}

__global__ void gSet(float* d_in, int* d_idx, int* index) {
  *index = d_idx[*index];
  d_in[*index] = -3.40282e+38f;
}

__global__ void gMaxElementUpdate(float* d_out, int* d_ind, float* d_in, int* blockID, int dist, int in_size) {
  extern __shared__ float sdata[];

  int tid = threadIdx.x;
  if (tid == 0) {
    d_in[d_ind[*blockID]] = -3.40282e+38f;
  }

  __syncthreads();
  __shared__ int indices[512];

  int i = *blockID * (blockDim.x * 2) + tid;

  sdata[tid] = -3.40282e+38f;

  if (i >= in_size) return;

  if (i + blockDim.x < in_size) {
    float a = d_in[i];
    float b = d_in[i+blockDim.x];
    if (a > b) {
      sdata[tid] = a;
      indices[tid] = i;
    } else {
      sdata[tid] = b;
      indices[tid] = i + blockDim.x;
    }
  } else {
    sdata[tid] = d_in[i];
    indices[tid] = i;
  }

  while (i + dist < in_size) {
    i += dist;

    float a = d_in[i];
    if (a > sdata[tid]) {
      sdata[tid] = a;
      indices[tid] = i;
    }

    if (i + blockDim.x < in_size) {
      float b = d_in[i + blockDim.x];
      if (b > sdata[tid]) {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }
  }
  i = *blockID * (blockDim.x * 2) + tid;

  __syncthreads();

  for (int s = (blockDim.x >> 1); s > 32; s >>= 1) {
    if (tid < s && tid + s < in_size) {
      if (sdata[tid + s] > sdata[tid]) {
        sdata[tid] = sdata[tid + s];
        indices[tid] = indices[tid + s];
      }
    }
    __syncthreads();
  }

  if (tid < 32 && tid + 32 < in_size) {
    if (sdata[tid + 32] > sdata[tid]) {
      sdata[tid] = sdata[tid + 32];
      indices[tid] = indices[tid + 32];
    }
  }

  if (tid < 16 && tid + 16 < in_size) {
    if (sdata[tid + 16] > sdata[tid]) {
      sdata[tid] = sdata[tid + 16];
      indices[tid] = indices[tid + 16];
    }
  }

  if (tid < 8 && tid + 8 < in_size) {
    if (sdata[tid + 8] > sdata[tid]) {
      sdata[tid] = sdata[tid + 8];
      indices[tid] = indices[tid + 8];
    }
  }

  if (tid < 4 && tid + 4 < in_size) {
    if (sdata[tid + 4] > sdata[tid]) {
      sdata[tid] = sdata[tid + 4];
      indices[tid] = indices[tid + 4];
    }
  }

  if (tid < 2 && tid + 2 < in_size) {
    if (sdata[tid + 2] > sdata[tid]) {
      sdata[tid] = sdata[tid + 2];
      indices[tid] = indices[tid + 2];
    }
  }

  if (tid < 1 && tid + 1 < in_size) {
    if (sdata[tid + 1] > sdata[tid]) {
      sdata[tid] = sdata[tid + 1];
      indices[tid] = indices[tid + 1];
    }
  }

  if (tid == 0) {
    d_out[*blockID] = sdata[0];
    int tmp = d_ind[*blockID];
    d_ind[*blockID] = indices[0];
    *blockID = tmp;
  }
}
__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
  int tid = threadIdx.x  + blockDim.x * blockIdx.x;
  if (tid < n) {
    int index = indeces[tid];
    d_out[tid] = d_in[index];
  }
}

NthElement::NthElement(size_t maxBeamSize, size_t maxBatchSize, hipStream_t& stream)
    : stream_(stream) {
  HANDLE_ERROR( hipMalloc((void**)&d_ind, BLOCK_SIZE * sizeof(int)) );

  HANDLE_ERROR( hipMalloc((void**)&d_out, BLOCK_SIZE * sizeof(float)) );

  HANDLE_ERROR( hipMalloc((void**)&d_res_idx, maxBatchSize * maxBeamSize * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&d_res, maxBatchSize * maxBeamSize * sizeof(float)) );

  hipHostAlloc((void**) &h_res, maxBeamSize * maxBatchSize* sizeof(float), hipHostMallocDefault);
  hipHostAlloc((void**) &h_res_idx, maxBeamSize * maxBatchSize * sizeof(int), hipHostMallocDefault);

  HANDLE_ERROR( hipMalloc((void**)&d_breakdown, maxBeamSize * sizeof(float)) );
}

void NthElement::getNBestList(float* d_in, size_t N, size_t n, size_t pos) {
  if (n == 0) return;

  const int N_BLOCKS = std::min(500, int(N / (2 * BLOCK_SIZE)) + int(N % (2 * BLOCK_SIZE) != 0));

  gMaxElement<<<N_BLOCKS, BLOCK_SIZE, BLOCK_SIZE * sizeof(float), stream_>>>
    (d_out, d_ind, d_in, N);

  for (size_t i = 0; i < n; ++i) {

    gMaxElement<<<1, 512, 512 * sizeof(float), stream_>>>
      (d_res + pos + i, d_res_idx + pos + i, d_out, N_BLOCKS);

    gMaxElementUpdate<<<1, BLOCK_SIZE, BLOCK_SIZE * sizeof(float), stream_>>>
      (d_out, d_ind, d_in, d_res_idx + pos + i, 2 * BLOCK_SIZE * N_BLOCKS, N);
  }
}

void NthElement::GetPairs(size_t number,
                    std::vector<unsigned>& outKeys,
                    std::vector<float>& outValues) {

  HANDLE_ERROR( hipMemcpyAsync(h_res, d_res, number * sizeof(float),
                                hipMemcpyDeviceToHost, stream_) );
  HANDLE_ERROR( hipMemcpyAsync(h_res_idx, d_res_idx, number * sizeof(int),
                                hipMemcpyDeviceToHost, stream_) );
  hipStreamSynchronize(stream_);

  for (size_t i = 0; i < number; ++i) {
    outKeys.push_back(h_res_idx[i]);
    outValues.push_back(h_res[i]);
  }

  lastN = number;
}

void NthElement::getValueByKey(std::vector<float>& out, float* d_in) {
  gGetValueByKey<<<1, lastN, 0, stream_>>>
    (d_in, d_breakdown, h_res_idx, lastN);

  HANDLE_ERROR( hipMemcpyAsync(out.data(), d_breakdown, lastN * sizeof(float),
                                hipMemcpyDeviceToHost, stream_) );
  hipStreamSynchronize(stream_);
}

}  // namespace GPU
