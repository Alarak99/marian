#include "hip/hip_runtime.h"
#include "gpu/decoder/nth_element.h"


namespace GPU {

static void HandleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void gMaxElement(float* d_out, int* d_ind, float* d_in, int in_size) {
  extern __shared__ float sdata[];
  __shared__ int indices[512];


  int tid = threadIdx.x;
  int i = blockIdx.x * (blockDim.x * 2) + tid;

  sdata[tid] = -3.40282e+38f;
  __syncthreads();

  if (i >= in_size) return;

  if (i + blockDim.x < in_size) {
    float a = d_in[i];
    float b = d_in[i+blockDim.x];
    if (a > b) {
      sdata[tid] = a;
      indices[tid] = i;
    } else {
      sdata[tid] = b;
      indices[tid] = i + blockDim.x;
    }
  } else {
    sdata[tid] = d_in[i];
    indices[tid] = i;
  }

  __syncthreads();

  for (int s = (blockDim.x >> 1); s > 0; s >>= 1) {
    if (tid < s) {
      if (sdata[tid + s] > sdata[tid]) {
        sdata[tid] = sdata[tid + s];
        indices[tid] = indices[tid + s];
      }
    }
    __syncthreads();
  }

  if (tid == 0) {
    d_out[blockIdx.x] = sdata[0];
    d_ind[blockIdx.x] = indices[0];
  }
}

__global__ void gSet(float* d_in, int* d_idx, int* index) {
  *index = d_idx[*index];
  // d_in[*index] = std::numeric_limits<float>::lowest();
  d_in[*index] = -3.40282e+38f;
}

NthElement::NthElement(size_t maxBeamSize, hipStream_t& stream)
    : stream_(stream) {
  HANDLE_ERROR( hipMalloc((void**)&d_ind, BLOCK_SIZE * sizeof(int)) );

  HANDLE_ERROR( hipMalloc((void**)&d_out, BLOCK_SIZE * sizeof(float)) );

  HANDLE_ERROR( hipMalloc((void**)&d_res_idx, maxBeamSize * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&d_res, maxBeamSize * sizeof(float)) );

  hipHostAlloc((void**) &h_res, maxBeamSize * sizeof(float), hipHostMallocDefault);
  hipHostAlloc((void**) &h_res_idx, maxBeamSize * sizeof(int), hipHostMallocDefault);
}

void NthElement::getNBestList(float* d_in, size_t N, size_t n,
                              std::vector<unsigned>& outKeys,
                              std::vector<float>& outValues) {
  if (n == 0) return;

  const int N_BLOCKS = (N / (2 * BLOCK_SIZE)) + 1;
  /* std::cerr << "#BLOCKS: " << N_BLOCKS << std::endl; */
  /* hipStreamSynchronize(stream_); */

  for (size_t i = 0; i < n; ++i) {
    gMaxElement<<<N_BLOCKS, BLOCK_SIZE, BLOCK_SIZE * sizeof(float), stream_>>>(d_out, d_ind, d_in, N);
    /* hipStreamSynchronize(stream_); */
    /* float *tmp= new float[N_BLOCKS]; */
    /* hipMemcpy(tmp, d_out, N_BLOCKS * sizeof(float), hipMemcpyDeviceToHost); */
    /* for (int k = 0; k < N_BLOCKS; ++k) std::cerr << k << ": " << tmp[k] << "\t"; */
    /* std::cerr << std::endl; */
    /* delete [] tmp; */
    gMaxElement<<<1, (512 / 2), (512 /2 ) * sizeof(float), stream_>>>(d_res + i, d_res_idx + i, d_out, N_BLOCKS);
    gSet<<<1, 1, 0, stream_>>>(d_in, d_ind, d_res_idx + i);
  }

  HANDLE_ERROR( hipMemcpyAsync(h_res, d_res, n * sizeof(float), hipMemcpyDeviceToHost, stream_) );
  HANDLE_ERROR( hipMemcpyAsync(h_res_idx, d_res_idx, n * sizeof(int), hipMemcpyDeviceToHost, stream_) );

  hipStreamSynchronize(stream_);

  for (size_t i = 0; i < n; ++i) {
    outKeys[i] = h_res_idx[i];
    outValues[i] = h_res[i];
    /* std::cerr << outKeys[i] << ": " << outValues[i] << "\t"; */
  }
  /* std::cerr << std::endl; */
}

}  // namespace GPU
