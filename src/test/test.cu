#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <boost/timer/timer.hpp>
#include <boost/algorithm/string.hpp>

#include "mblas/matrix.h"
#include "bahdanau.h"
#include "vocab.h"

#include "states.h"

using namespace mblas;

int main(int argc, char** argv) {
  size_t device = 0;
  
  if(argc > 1) {
    if(std::string(argv[1]) == "1")
      device = 1;
    else if(std::string(argv[1]) == "2")
      device = 2;
  }
  
  std::cerr << device << std::endl;
  hipSetDevice(device);
  
  std::string source = "thank you .";
  std::string target = "vielen dank .";
  //std::string source = "you know , one of the intense pleasures of travel and one of the delights of ethnographic research is the opportunity to live amongst those who have not forgotten the old ways , who still feel their past in the wind , touch it in stones polished by rain , taste it in the bitter leaves of plants .";
  //std::string target = "wissen sie , eine der intensiven freuden des reisens und eine der freuden der ethnografischen forschung ist die chance zu leben unter jenen , die die alten wege nicht vergessen haben , die immer noch ihre vergangenheit im wind spüren , berühren sie in steine poliert durch regen , schmecken sie in den bitteren blätter der pflanzen .";
  
  std::cerr << "Loading model" << std::endl;
  Weights weights("/home/marcinj/Badania/best_nmt/search_model.npz", device);
  Vocab svcb("/home/marcinj/Badania/best_nmt/vocab/en_de.en.txt");
  Vocab tvcb("/home/marcinj/Badania/best_nmt/vocab/en_de.de.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);
  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
  
  std::vector<std::string> sourceSplit;
  boost::split(sourceSplit, source, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Source: " << std::endl;
  std::vector<size_t> sWords(sourceSplit.size());
  std::transform(sourceSplit.begin(), sourceSplit.end(), sWords.begin(),
                 [&](const std::string& w) { std::cerr << svcb[w] << ", "; return svcb[w]; });
  sWords.push_back(svcb["</s>"]);
  std::cerr << svcb["</s>"] << std::endl;
  
  typedef std::vector<size_t> Batch;
  
  std::vector<std::string> targetSplit;
  boost::split(targetSplit, target, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Target: " << std::endl;
  size_t bs = 1000;
  std::vector<std::vector<size_t>> tWordsBatch(targetSplit.size());
  std::transform(targetSplit.begin(), targetSplit.end(), tWordsBatch.begin(),
                 [&](const std::string& w) { std::cerr << tvcb[w] << ", "; return Batch(bs, tvcb[w]); });
  tWordsBatch.push_back(Batch(bs, tvcb["</s>"]));
  std::cerr << tvcb["</s>"] << std::endl;

  mblas::Matrix SourceContext;
  encoder.GetContext(sWords, SourceContext);

  mblas::Matrix State, NextState;
  mblas::Matrix Embeddings, NextEmbeddings;
  mblas::Matrix Probs;

  std::cerr << "Testing" << std::endl;
  boost::timer::auto_cpu_timer timer;
  size_t batchSize = tWordsBatch[0].size();
  
  for(size_t i = 0; i < 1; ++i) {
    decoder.EmptyState(State, SourceContext, batchSize);
    decoder.EmptyEmbedding(Embeddings, batchSize);
    
    float sum = 0;
    for(auto batch : tWordsBatch) {
      decoder.MakeStep(NextState, NextEmbeddings, Probs,
                       batch, State, Embeddings, SourceContext);

      for(size_t i = 0; i < 1; ++i) {
        float p = Probs(i, batch[i]);
        std:: cerr << log(p) << " ";
        if(i == 0) {
          sum += log(p);
        }
      }

      mblas::Swap(Embeddings, NextEmbeddings);
      mblas::Swap(State, NextState);
    }
    std::cerr << i << " " << sum << std::endl;
  }
}
