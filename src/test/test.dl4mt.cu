#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <boost/timer/timer.hpp>
#include <boost/algorithm/string.hpp>

#include "mblas/matrix.h"
#include "dl4mt.h"
#include "vocab.h"

using namespace mblas;

int main(int argc, char** argv) {
  size_t device = 0;
  
  if(argc > 1) {
    if(std::string(argv[1]) == "1")
      device = 1;
    else if(std::string(argv[1]) == "2")
      device = 2;
  }
  
  std::cerr << device << std::endl;
  hipSetDevice(device);
  
  std::string source = "thank you .";
  std::string target = "vielen Dank .";
  
  std::cerr << "Loading model" << std::endl;
  Weights weights("testmodel/model.npz", device);
  
  Vocab svcb("testmodel/vocab.en.txt");
  Vocab tvcb("testmodel/vocab.de.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);

  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
    
  std::vector<std::string> sourceSplit;
  boost::split(sourceSplit, source, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Source: " << std::endl;
  std::vector<size_t> sWords(sourceSplit.size());
  std::transform(sourceSplit.begin(), sourceSplit.end(), sWords.begin(),
                 [&](const std::string& w) { std::cerr << svcb[w] << ", "; return svcb[w]; });
  sWords.push_back(svcb["</s>"]);
  std::cerr << svcb["</s>"] << std::endl;
  
  typedef std::vector<size_t> Batch;
  
  std::vector<std::string> targetSplit;
  boost::split(targetSplit, target, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Target: " << std::endl;
  size_t bs = 1000;
  
  std::vector<std::vector<size_t>> tWordsBatch(targetSplit.size());
  std::transform(targetSplit.begin(), targetSplit.end(), tWordsBatch.begin(),
                 [&](const std::string& w) { std::cerr << tvcb[w] << ", "; return Batch(bs, tvcb[w]); });
  tWordsBatch.push_back(Batch(bs, tvcb["</s>"]));
  std::cerr << tvcb["</s>"] << std::endl;

  mblas::Matrix SourceContext;
  encoder.GetContext(sWords, SourceContext);

  mblas::Matrix State, NextState;
  mblas::Matrix Embeddings, NextEmbeddings;
  mblas::Matrix Probs;

  std::cerr << "Testing" << std::endl;
  boost::timer::auto_cpu_timer timer;
  size_t batchSize = tWordsBatch[0].size();

  for(size_t i = 0; i < 1; ++i) {
    decoder.EmptyState(State, SourceContext, batchSize);
    decoder.EmptyEmbedding(Embeddings, batchSize);
    
    float sum = 0;
    for(auto batch : tWordsBatch) {
      decoder.MakeStep(NextState, Probs,
                       State, Embeddings, SourceContext);
      decoder.Lookup(NextEmbeddings, batch);
      for(size_t i = 0; i < 1; ++i) {
        float p = Probs(i, batch[i]);
        if(i == 0) {
          sum += log(p);
        }
      }
      
      mblas::Swap(Embeddings, NextEmbeddings);
      mblas::Swap(State, NextState);
    }
    std::cerr << i << " " << sum << std::endl;
  }
}
