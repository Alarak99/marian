#include <cstdlib>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <limits> 
#include <boost/timer/timer.hpp>
#include <boost/algorithm/string.hpp>

#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>


#include "mblas/matrix.h"
#include "dl4mt.h"
#include "vocab.h"

using namespace mblas;

typedef std::tuple<size_t, size_t, float> Hypothesis;
typedef std::vector<Hypothesis> Beam;
typedef std::vector<Beam> History;

void BestHyps(Beam& bestHyps, const Beam& prevHyps, mblas::Matrix& Probs, const size_t beamSize) {
  mblas::Matrix Costs(Probs.Rows(), 1);
  thrust::host_vector<float> vCosts;
  for(const Hypothesis& h : prevHyps)
    vCosts.push_back(std::get<2>(h));
  thrust::copy(vCosts.begin(), vCosts.end(), Costs.begin());
  
  mblas::BroadcastVecColumn(Log(_1) + _2, Probs, Costs);
  
  thrust::device_vector<unsigned> keys(Probs.size());
  thrust::sequence(keys.begin(), keys.end());
  
  // Here it would be nice to have a partial sort instead of full sort
  thrust::sort_by_key(Probs.begin(), Probs.end(),
                      keys.begin(), thrust::greater<float>());
  
  thrust::host_vector<unsigned> bestKeys(beamSize);
  thrust::copy_n(keys.begin(), beamSize, bestKeys.begin());
  thrust::host_vector<float> bestCosts(beamSize);
  thrust::copy_n(Probs.begin(), beamSize, bestCosts.begin());
  
  for(size_t i = 0; i < beamSize; i++) {
    size_t wordIndex = bestKeys[i] % Probs.Cols();
    size_t hypIndex  = bestKeys[i] / Probs.Cols();
    float  cost = bestCosts[i];
    bestHyps.emplace_back(wordIndex, hypIndex, cost);  
  }
}

void FindBest(const History& history, const Vocab& vcb) {
  std::vector<size_t> targetWords;
  
  size_t best = 0;
  size_t beamSize = 0;
  float bestCost = std::numeric_limits<float>::lowest();
      
  for(auto b = history.rbegin(); b != history.rend(); b++) {
    if(b->size() > beamSize) {
      beamSize = b->size();
      for(size_t i = 0; i < beamSize; ++i) {
        if(b == history.rbegin() || std::get<0>((*b)[i]) == vcb["</s>"]) {
          if(std::get<2>((*b)[i]) > bestCost) {
            best = i;
            bestCost = std::get<2>((*b)[i]);
            targetWords.clear();
          }
        }
      }
    }
    
    auto& bestHyp = (*b)[best];
    targetWords.push_back(std::get<0>(bestHyp));
    best = std::get<1>(bestHyp);
  }

  std::reverse(targetWords.begin(), targetWords.end());
  for(size_t i = 0; i < targetWords.size(); ++i) {
    if(vcb[targetWords[i]] != "</s>") {
      if(i > 0) {
        std::cout << " ";
      }
      std::cout << vcb[targetWords[i]];
    }
  }
  std::cout << std::endl;
}

int main(int argc, char** argv) {
  size_t device = 0;
  
  if(argc > 1) {
    if(std::string(argv[1]) == "1")
      device = 1;
    else if(std::string(argv[1]) == "2")
      device = 2;
  }
  
  hipSetDevice(device);
  
  Weights weights("/home/marcinj/Badania/mosesNMT/moses/FF/NMT/testmodel/model.npz", device);
  Vocab svcb("/home/marcinj/Badania/mosesNMT/moses/FF/NMT/testmodel/vocab.en.txt");
  Vocab tvcb("/home/marcinj/Badania/mosesNMT/moses/FF/NMT/testmodel/vocab.de.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);

  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
  
  mblas::Matrix State, NextState, BeamState;
  mblas::Matrix Embeddings, NextEmbeddings;
  mblas::Matrix Probs;

  std::string source;
  boost::timer::auto_cpu_timer timer;
  
  while(std::getline(std::cin, source)) {
    std::vector<std::string> sourceSplit;
    boost::split(sourceSplit, source, boost::is_any_of(" "),
                 boost::token_compress_on);
      
    std::vector<size_t> sourceWords(sourceSplit.size());
    std::transform(sourceSplit.begin(), sourceSplit.end(), sourceWords.begin(),
                   [&](const std::string& w) { return svcb[w]; });
    sourceWords.push_back(svcb["</s>"]);
    
    mblas::Matrix SourceContext;
    encoder.GetContext(sourceWords, SourceContext);
  
    size_t beamSize = 12;
    
    decoder.EmptyState(State, SourceContext, 1);
    decoder.EmptyEmbedding(Embeddings, 1);
    
    History history;
    
    Beam prevHyps;
    prevHyps.emplace_back(0, 0, 0.0);
    
    do {
      decoder.MakeStep(NextState, Probs, State, Embeddings, SourceContext);
      
      Beam hyps;
      BestHyps(hyps, prevHyps, Probs, beamSize);
      history.push_back(hyps);
      
      Beam survivors;
      std::vector<size_t> beamWords;
      std::vector<size_t> beamStateIds;
      for(auto& h : hyps) {
        if(std::get<0>(h) != tvcb["</s>"]) {
          survivors.push_back(h);
          beamWords.push_back(std::get<0>(h));
          beamStateIds.push_back(std::get<1>(h));
        }
      }
      beamSize = survivors.size();
      
      if(beamSize == 0)
        break;
      
      decoder.Lookup(NextEmbeddings, beamWords);
      mblas::Assemble(BeamState, NextState, beamStateIds);
      
      mblas::Swap(Embeddings, NextEmbeddings);
      mblas::Swap(State, BeamState);
      prevHyps.swap(survivors);
      
    } while(history.size() < sourceWords.size() * 3);
    
    FindBest(history, tvcb);
  }
}