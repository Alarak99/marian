#include <cstdlib>
#include <iostream>
#include <string>
#include <algorithm>
#include <memory>
#include <boost/timer/timer.hpp>
#include <boost/program_options/options_description.hpp>
#include <boost/program_options/parsers.hpp>
#include <boost/program_options/variables_map.hpp>
#include <boost/lexical_cast.hpp>

#include "dl4mt.h"
#include "vocab.h"
#include "search.h"

class BPE {
  public:
    BPE(const std::string& sep = "@@ ")
     : sep_(sep) {}
    
    std::string split(const std::string& line) {
      return line;
    }
    
    std::string unsplit(const std::string& line) {
      std::string joined = line;
      size_t pos = joined.find(sep_);
      while(pos != std::string::npos) {
        joined.erase(pos, sep_.size());
        pos = joined.find(sep_, pos);
      }
      return joined;
    }
    
    operator bool() const {
      return true;
    }
    
  private:
    std::string sep_;
};

int main(int argc, char* argv[]) {
  std::string srcVocabPath, trgVocabPath;
  std::vector<std::string> modelPaths;
  size_t device = 0;
  size_t nbest = 0;
  size_t beamSize = 12;
  bool help = false;

  namespace po = boost::program_options;
  po::options_description cmdline_options("Allowed options");
  cmdline_options.add_options()
    ("beamsize,b", po::value(&beamSize)->default_value(12),
     "Beam size")
    ("n-best-list", po::value(&nbest)->default_value(0),
     "N-best list")
    ("device,d", po::value(&device)->default_value(0),
     "CUDA Device")
    ("model(s),m", po::value(&modelPaths)->multitoken()->required(),
     "Path to a model")
    ("source,s", po::value(&srcVocabPath)->required(),
     "Path to a source vocab file.")
    ("target,t", po::value(&trgVocabPath)->required(),
     "Path to a target vocab file.")
    ("help,h", po::value(&help)->zero_tokens()->default_value(false),
     "Print this help message and exit.");
  
  po::variables_map vm;
  try {
    po::store(po::command_line_parser(argc, argv).
              options(cmdline_options).run(), vm);
    po::notify(vm);
  } catch (std::exception& e) {
    std::cout << "Error: " << e.what() << std::endl << std::endl;

    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }

  if (help) {
    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }

  std::cerr << "Using device GPU" << device << std::endl;;
  hipSetDevice(device);
  Vocab srcVocab(srcVocabPath);
  Vocab trgVocab(trgVocabPath);
  
  std::vector<std::unique_ptr<Weights>> models;
  for(auto& modelPath : modelPaths) {
    std::cerr << "Loading model " << modelPath << std::endl;
    models.emplace_back(new Weights(modelPath));
  }
  
  std::cerr << "done." << std::endl;

  Search search(models, nbest > 0);

  std::cerr << "Translating...\n";

  std::ios_base::sync_with_stdio(false);

  BPE bpe;
  
  boost::timer::cpu_timer timer;
  std::string in;
  size_t lineCounter = 0;
  while(std::getline(std::cin, in)) {
    Sentence sentence = bpe ? srcVocab(bpe.split(in)) : srcVocab(in);
    History history = search.Decode(sentence, beamSize);
    std::string out = trgVocab(history.Top().first);
    if(bpe)
      out = bpe.unsplit(out);
    std::cout << out << std::endl;
    if(nbest > 0) {
      NBestList nbl = history.NBest(beamSize);
      for(size_t i = 0; i < nbl.size(); ++i) {
        auto& r = nbl[i];
        std::cout << lineCounter << " ||| " << bpe.unsplit(trgVocab(r.first)) << " |||";
        for(size_t j = 0; j < r.second.GetCostBreakdown().size(); ++j) {
          std::cout << " F" << j << "=" << r.second.GetCostBreakdown()[j];
        }
        std::cout << " ||| " << r.second.GetCost() << std::endl;
      }
    }
    lineCounter++;
  }
  std::cerr << timer.format() << std::endl;
  return 0;
}
