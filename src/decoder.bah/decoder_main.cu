#include <cstdlib>
#include <iostream>
#include <string>
#include <algorithm>
#include <memory>
#include <boost/timer/timer.hpp>
#include <boost/program_options/options_description.hpp>
#include <boost/program_options/parsers.hpp>
#include <boost/program_options/variables_map.hpp>
#include <boost/lexical_cast.hpp>

#include "bahdanau/model.h"
#include "vocab.h"
#include "decoder/nmt_decoder.h"


void ProgramOptions(int argc, char *argv[],
    std::string& modelPath,
    std::string& svPath,
    std::string& tvPath,
    size_t& beamsize,
    size_t& device) {
  bool help = false;

  namespace po = boost::program_options;
  po::options_description cmdline_options("Allowed options");
  cmdline_options.add_options()
    ("beamsize,b", po::value(&beamsize)->default_value(10),
     "Beam size")
    ("device,d", po::value(&device)->default_value(0),
     "CUDA Device")
    ("model,m", po::value(&modelPath)->required(),
     "Path to a model")
    ("source,s", po::value(&svPath)->required(),
     "Path to a source vocab file.")
    ("target,t", po::value(&tvPath)->required(),
     "Path to a target vocab file.")
    ("help,h", po::value(&help)->zero_tokens()->default_value(false),
     "Print this help message and exit.")
  ;
  po::variables_map vm;
  try {
    po::store(po::command_line_parser(argc, argv).
              options(cmdline_options).run(), vm);
    po::notify(vm);
  } catch (std::exception& e) {
    std::cout << "Error: " << e.what() << std::endl << std::endl;

    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }

  if (help) {
    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }
}

int main(int argc, char* argv[]) {
  std::string modelPath, srcVocabPath, trgVocabPath;
  size_t device = 0;
  size_t beamsize = 10;
  ProgramOptions(argc, argv, modelPath, srcVocabPath, trgVocabPath, beamsize, device);
  std::cerr << "Using device GPU" << device << std::endl;;
  hipSetDevice(device);
  std::cerr << "Loading model... ";
  std::shared_ptr<Weights> model(new Weights(modelPath));
  std::shared_ptr<Vocab> srcVocab(new Vocab(srcVocabPath));
  std::shared_ptr<Vocab> trgVocab(new Vocab(trgVocabPath));
  std::cerr << "done." << std::endl;

  NMTDecoder decoder(model, srcVocab, trgVocab, beamsize);

  std::cerr << "Start translating...\n";

  std::ios_base::sync_with_stdio(false);

  std::string line;
  boost::timer::cpu_timer timer;
  while(std::getline(std::cin, line)) {
    auto result = decoder.translate(line);
    for (auto it = result.rbegin(); it != result.rend(); ++it) {
      std::string word = (*trgVocab)[*it];
      if(it != result.rbegin())
        std::cout << " ";
      if(word != "</s>")
        std::cout << word;
    }
    std::cout << std::endl;
  }
  std::cerr << timer.format() << std::endl;
  return 0;
}
