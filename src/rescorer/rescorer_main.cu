#include <iostream>
#include <string>
#include <memory>
#include <iomanip>

#include <boost/program_options/options_description.hpp>
#include <boost/program_options/parsers.hpp>
#include <boost/program_options/variables_map.hpp>
#include <boost/timer/timer.hpp>
#include <boost/chrono/duration.hpp>

typedef boost::chrono::duration<double> sec;

#include "nbest.h"
#include "vocab.h"
#include "rescorer.h"

void ProgramOptions(int argc, char *argv[],
    std::string& modelPath,
    std::string& svPath,
    std::string& tvPath,
    std::string& corpusPath,
    std::string& nbestPath,
    std::string& fname,
    size_t& maxBatchSize,
    size_t& device) {
  bool help = false;

  namespace po = boost::program_options;
  po::options_description cmdline_options("Allowed options");
  cmdline_options.add_options()
    ("device,d", po::value(&device)->default_value(0),
     "CUDA Device")
    ("batch,b", po::value(&maxBatchSize)->default_value(80),
     "Max batch size")
    ("model,m", po::value(&modelPath)->required(),
     "Path to a model")
    ("source,s", po::value(&svPath)->required(),
     "Path to a source vocab file.")
    ("target,t", po::value(&tvPath)->required(),
     "Path to a target vocab file.")
    ("input,i", po::value(&corpusPath)->required(),
     "Path to the input of the nbest file.")
    ("n-best,n", po::value(&nbestPath)->required(),
     "Path to an nbest file.")
    ("feature-name,f", po::value(&fname)->default_value("NMT0"),
     "Feature name")
    ("help,h", po::value(&help)->zero_tokens()->default_value(false),
     "Print this help message and exit.")
  ;
  po::variables_map vm;
  try {
    po::store(po::command_line_parser(argc, argv).
              options(cmdline_options).run(), vm);
    po::notify(vm);
  } catch (std::exception& e) {
    std::cout << "Error: " << e.what() << std::endl << std::endl;

    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }

  if (help) {
    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }
}

int main(int argc, char* argv[]) {
  std::string modelPath, svPath, tvPath, corpusPath, nbestPath, fname;

  size_t device;
  size_t maxBatchSize;
  ProgramOptions(argc, argv, modelPath, svPath,tvPath, corpusPath, nbestPath,
                 fname, maxBatchSize, device);
  std::cerr << "Using device: " << device << std::endl;
  hipSetDevice(device);

  std::cerr << "Loading model: " << modelPath << std::endl;
  std::shared_ptr<Weights> weights(new Weights(modelPath, device));
  std::shared_ptr<Vocab> srcVocab(new Vocab(svPath));
  std::shared_ptr<Vocab> trgVocab(new Vocab(tvPath));

  std::cerr << "Loading nbest list: " << nbestPath << std::endl;
  std::shared_ptr<NBest> nbest(new NBest(corpusPath,nbestPath, srcVocab, trgVocab, maxBatchSize));

  std::cerr << "Creating rescorer..." << std::endl;
  std::shared_ptr<Rescorer> rescorer(new Rescorer(weights, nbest, fname));

  boost::timer::cpu_timer timer;
  std::cerr << std::fixed << std::setprecision(2);
  std::cerr << "Rescoring..." << std::endl;
  for (size_t i = 0; i < nbest->size(); ++i) {
    rescorer->Score(i);
    std::cerr << ".";
    if((i + 1) % 50 == 0) {
        sec seconds = boost::chrono::nanoseconds(timer.elapsed().user);
        std::cerr << "[" << i + 1 << " / " << seconds.count() << " s = " << (i+1)/seconds.count() << " sent. per s]" << std::endl;
    }
  }

  return 0;
}
