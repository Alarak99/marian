#include "hip/hip_runtime.h"
#include "gpu/mblas/matrix_functions.h"
#include "gpu/mblas/handles.h"

using namespace std;

namespace amunmt {
namespace GPU {
namespace mblas {

thread_local hipblasHandle_t* CublasHandler::handle_ = nullptr;
thread_local CudaStreamHandler* CudaStreamHandler::instance_ = nullptr;;

Matrix& Swap(Matrix& Out, Matrix& In) {
  Out.swap(In);
  return Out;
}

__global__ void gMean(MatrixWrapper<float> out,
                      const MatrixWrapper<float> in,
                      const MatrixWrapper<int>  mapping)
{
  assert(out.dim(0) == 1);
  // in = max sentence length, whatever, 1, batches
  // out = in, dim(0 = 1
  // mapping = max length * batches

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  //printf("id = %d in = %lu %lu %lu %lu = %lu %lu \n", id, in.dim(0), in.dim(1), in.dim(2), in.dim(3), in.size(), sizeof(in));

  if (id < out.size()) {
    size_t indices[SHAPE_SIZE];
    out.id2Indices(id, indices);
    assert(indices[0] == 0);
    //printf("%d -> %lu %lu %lu %lu \n", id, indices[0], indices[1], indices[2], indices[3]);

    size_t batch = indices[3];
    size_t startMapInd = batch * in.dim(0);

    float sum = 0.0f;
    int counter = 0;
    for (size_t row = 0; row < in.dim(0); ++row) {
      int isWord = mapping(row, batch, 0, 0);
      //printf("batch=%lu startMapInd=%lu  mapOffset=%lu -> %d \n", batch, startMapInd, mapOffset, isWord);
      if (isWord) {
        sum += in(row, indices[1], indices[2], indices[3]);
        ++counter;
      }
    }

    sum /= (float) counter;
    out[id] = sum;
  }
}

void Mean(Matrix& Out, const Matrix& In, const DeviceVector<int>& mapping) {
  size_t batchNum = Out.dim(0) * Out.dim(2) * Out.dim(3);
  size_t stateLength = Out.dim(1);
  size_t sentenceLength = (In.dim(0) * In.dim(2) * In.dim(3)) / batchNum;

  MatrixWrapper<float> outWrap(Out);
  MatrixWrapper<float> inWrap(In);

  MatrixWrapper<int> mappingWrap(mapping, sentenceLength, batchNum, 1, 1);

  size_t threads = MAX_THREADS;
  size_t blocks =  (outWrap.size() / threads) + ((outWrap.size() % threads == 0) ?  0 : 1);

  gMean<<<blocks, threads, 0, CudaStreamHandler::GetStream()>>>
    (outWrap, inWrap, mappingWrap);

}

__global__ void gWeightedMeanOld(float* d_out, const float* weights, const float* d_in, const int* mapping,
                              int numRows, int numCols, int srcLen) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < numRows * numCols) {
    int rowNo = id / numCols;
    int batchNo = mapping[rowNo];
    int statePos = id % numCols;

    float sum = 0.0f;
    for (int i = 0; i < srcLen; ++i) {
      sum += weights[rowNo * srcLen + i] * d_in[batchNo * srcLen * numCols + (i * numCols) + statePos];
    }

    d_out[id] = sum;
  }
}


__global__ void gWeightedMean(MatrixWrapper<float> out,
                              const MatrixWrapper<float> weights,
                              const MatrixWrapper<float> in,
                              const MatrixWrapper<int> mapping
                              )
{
  int numHypos = weights.dim(0);
  int states = in.dim(1);
  int srcLen = weights.dim(1);

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < numHypos * states) {
    int hypoInd = id / states;
    int batchInd = mapping[hypoInd];
    int stateInd = id % states;
    //printf("hypoInd=%d batchInd=%d stateInd=%d \n", hypoInd, batchInd, stateInd);

    float sum = 0.0f;
    for (uint i = 0; i < srcLen; ++i) {
      sum += weights(hypoInd, i, 0, 0) * in(i, stateInd, 0, batchInd);
    }

    out[id] = sum;
  }
}

void WeightedMean(Matrix& Out,const Matrix& Weights, const Matrix& In, const DeviceVector<int>& mapping) {
  int numHypos = Weights.dim(0);
  int states = In.dim(1);

  Out.Resize(numHypos, states);

  MatrixWrapper<float> outWrap(Out);
  MatrixWrapper<float> weightsWrap(Weights);
  MatrixWrapper<float> inWrap(In);
  MatrixWrapper<int> mappingWrap(mapping);

  int nThreads = MAX_THREADS;
  int nBlocks =  (Out.size() / MAX_THREADS) + ((Out.size() % MAX_THREADS == 0) ?  0 : 1);

  gWeightedMean<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
    (outWrap, weightsWrap, inWrap, mappingWrap);
  /*
  cerr << "nBlocks=" << nBlocks << endl;

  cerr << "Out=" << outWrap.Debug() << endl;
  cerr << "Weights=" << weightsWrap.Debug() << endl;
  cerr << "In=" << inWrap.Debug() << endl;
  cerr << "mapping=" << mapping.size() << endl;
  for (size_t i = 0; i < mapping.size(); ++i) {
    cerr << mapping[i] << " ";
  }
  cerr << endl << endl;
  */
}

Matrix& Transpose(Matrix& Out, const Matrix& In) {
  size_t m = In.dim(0);
  size_t n = In.dim(1);

  Out.Resize(n, m);

  float alpha = 1.0;
  float beta  = 0.0;

  hipblasSgeam(CublasHandler::GetHandle(), HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, In.data(), n,
              &beta, In.data(), n, Out.data(), m);

  return Out;
}

Matrix& Transpose(Matrix& Out) {
  Matrix Temp;
  Transpose(Temp, Out);
  Swap(Out, Temp);
  return Out;
}

Matrix& Concat(Matrix& Out, const Matrix& In) {
  size_t oldSize = Out.size();
  Out.Resize(Out.dim(0) + In.dim(0), Out.dim(1));

  mblas::copy(In.data(), In.size(), Out.data() + oldSize, hipMemcpyDeviceToDevice);

  return Out;
}

Matrix& Copy(Matrix& Out, const Matrix& In) {
  Out.Resize(In.dim(0), In.dim(1), In.dim(2), In.dim(3));

  mblas::copy(In.data(), In.size(), Out.data(), hipMemcpyDeviceToDevice);

  return Out;
}

__global__ void gPasteRows(float* d_out, int outRows, int outCols, const float* d_in, int inRows, int inCols, int colNo, int sparse) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < inRows * inCols) {
    int inRow = id / inCols;
    int inCol = id % inCols;
    int outID = (outRows + sparse * inRow) * outCols + inCol + colNo;
    d_out[outID] = d_in[id];
  }
}
void PasteRows(Matrix& Out, const Matrix& In, const size_t rowNo, size_t colNo, size_t sparse) {
  int nColumns = In.dim(1);
  int nRows = In.dim(0);
  int nThreads = 512;
  int nBlocks =  (In.size() / 512) + ((In.size() % 512 == 0) ?  0 : 1);

  /*
  cerr << "1Out=" << Out.Debug(1) << endl;
  cerr << "In=" << In.Debug(1) << endl;
  cerr << "rowNo=" << rowNo << endl;
  cerr << "colNo=" << colNo << endl;
  cerr << "sparse=" << sparse << endl;
  */
  gPasteRows<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
    (Out.data(), rowNo, Out.dim(1), In.data(), In.dim(0), In.dim(1), colNo, sparse);

  //cerr << "2Out=" << Out.Debug(1) << endl;

}

Matrix& PasteRow(Matrix& Out,
                 const Matrix& In,
                 const size_t r, const size_t c) {
  size_t start = r * Out.dim(1) + c;

  mblas::copy(In.data(), In.size(), Out.data() + start, hipMemcpyDeviceToDevice);

  return Out;
}

Matrix& CopyRow(Matrix& Out,
                const Matrix& In,
                const size_t r, const size_t c) {
  size_t length = In.dim(1) - c;
  Out.Resize(1, length);
  size_t start = r * In.dim(1) + c;
  //size_t end   = start + length;

  //mblas::copy(In.begin() + start, In.begin() + end, Out.begin());
  mblas::copy(In.data() + start, length , Out.data(), hipMemcpyDeviceToDevice);

  return Out;
}

__global__ void gCopyRows(float* out, const float* in, size_t cols,
                          const size_t* targetRowIdx, size_t numPairs) {
  for (int bid = 0; bid < numPairs; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if (j < numPairs) {
      size_t dstId = j;
      size_t srcId = targetRowIdx[j];

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

Matrix& CopyRows(Matrix& Out,
                 const Matrix& In,
                 const DeviceVector<size_t>& indices)
{
  float* d_out = Out.data();
  const float* d_in = In.data();

  const size_t* dev = thrust::raw_pointer_cast(indices.data());
  size_t numPairs = indices.size();

  int threads = std::min(MAX_THREADS, (int)In.dim(1));
  int blocks = std::min(MAX_BLOCKS, (int)numPairs);

  /*
  cerr << "Out=" << Out.Debug() << endl;
  cerr << "In=" << In.Debug() << endl;
  cerr << "cols=" << In.dim(1) << endl;

  cerr << "dev=" << dev << ": ";
  for (size_t i = 0; i < numPairs; ++i) {
    cerr << indices[i] << " ";
  }
  cerr << endl;
  cerr << "numPairs=" << numPairs << endl;
  */

  gCopyRows<<<blocks, threads, 0, CudaStreamHandler::GetStream()>>>
    (d_out, d_in, In.dim(1), dev, numPairs);

  return Out;
}


Matrix& Assemble(Matrix& Out,
                 const Matrix& In,
                 const DeviceVector<size_t>& indices) {
  Out.Resize(indices.size(), In.dim(1));
  //cerr << "Assemble=" << Out.Debug() << " " << In.Debug() << indices.size() << endl;

  CopyRows(Out, In, indices);
  return Out;
}

__global__ void gSlice(float* out, const float* in,
                       size_t n, size_t dim,
                       size_t rows, size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * dim;
      const float* rowIn = in + j * cols + n * dim;

      for(int tid = 0; tid < dim; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < dim)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

Matrix& Slice(Matrix& Out,
              const Matrix& In,
              size_t n, size_t dim) {

  Out.Resize(In.dim(0), dim);

  float* d_out = Out.data();
  const float* d_in = In.data();

  int threads = std::min(MAX_THREADS, (int)dim);
  int blocks = std::min(MAX_BLOCKS, (int)In.dim(0));

  gSlice<<<blocks, threads, 0, CudaStreamHandler::GetStream()>>>
    (d_out, d_in, n, dim, In.dim(0), In.dim(1));
  return Out;
}

Matrix& Prod(hipblasHandle_t handle, Matrix& C, const Matrix& A, const Matrix& B,
             bool transA, bool transB) {
  Matrix::value_type alpha = 1.0;
  Matrix::value_type beta = 0.0;

  size_t m = A.dim(0);
  size_t k = A.dim(1);
  if(transA)
    std::swap(m, k);

  size_t l = B.dim(0);
  size_t n = B.dim(1);
  if(transB)
    std::swap(l, n);

  size_t lda = A.dim(1);
  size_t ldb = B.dim(1);
  size_t ldc = B.dim(1);

  if(transB)
    ldc = B.dim(0);

  C.Resize(m, n, A.dim(2), A.dim(3));
  //cerr << "C=" << C.Debug(1) << endl;

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  size_t m2 = A.dim(0) * A.dim(2) * A.dim(3);

  hipblasSgemm(handle, opB, opA,
              n, m2, k, &alpha, B.data(), ldb, A.data(), lda, &beta, C.data(), ldc);
  return C;
}

Matrix& Prod(Matrix& C, const Matrix& A, const Matrix& B,
             bool transA, bool transB) {

  //std::cerr << "1C=" << C.Debug() << std::endl;
  //std::cerr << "1A=" << A.Debug() << std::endl;
  //std::cerr << "1B=" << B.Debug() << std::endl;

  Matrix &ret = Prod(CublasHandler::GetHandle(), C, A, B, transA, transB);

  //std::cerr << "2C=" << C.Debug() << std::endl;
  return ret;
}

__global__ void gSoftMax(float* softMaxP, size_t rows, size_t cols,
                         const int* batchID,
                         int batchNum,
                         const int* srcMapping,
                         int srcNum) {
  extern __shared__ float _share[];

  int rowIdx =  blockIdx.x;

  while (rowIdx < rows) {
    float* row = softMaxP + rowIdx * cols;

    float* _max = _share;
    _max[threadIdx.x] = row[threadIdx.x];
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        float value = row[id];
        value *= srcMapping[ batchID[rowIdx] * srcNum + id ];
        if (value > _max[threadIdx.x]) {
          _max[threadIdx.x] = value;
        }
      }
    }

    int len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        if(_max[threadIdx.x + skip] > _max[threadIdx.x])
          _max[threadIdx.x] = _max[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }
    __syncthreads();
    float max = _max[0];
    __syncthreads();

    float* _sum = _share;// + blockDim.x;
    _sum[threadIdx.x] = 0.0f;
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        row[id] = __expf(row[id] - max);
        row[id] *= srcMapping[ batchID[rowIdx] * srcNum + id ];
        _sum[threadIdx.x] += row[id];
      }
    }

    __syncthreads();

    len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        _sum[threadIdx.x] += _sum[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }

    __syncthreads();

    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        row[id] /= _sum[0];
      }
    }
    __syncthreads();
    rowIdx += gridDim.x;
  }
}

Matrix& Softmax(Matrix& Out, const DeviceVector<int>& batchIds, const DeviceVector<int>& srcMapping,size_t srcSize) {
  int blocks = std::min(MAX_BLOCKS, (int)Out.dim(0));
  int threads = std::min(MAX_THREADS, (int)Out.dim(1));
  int shared = sizeof(float) * threads * 2;

  gSoftMax<<<blocks, threads, shared, CudaStreamHandler::GetStream()>>>
    (Out.data(), Out.dim(0), Out.dim(1),
     thrust::raw_pointer_cast(batchIds.data()), batchIds.size(),
     thrust::raw_pointer_cast(srcMapping.data()), srcSize);
  return Out;
}

__global__ void gLogSoftMax(float* softMaxP, size_t rows, size_t cols) {
  extern __shared__ float _share[];

  int rowIdx =  blockIdx.x;

  while (rowIdx < rows) {
    float* row = softMaxP + rowIdx * cols;

    float* _max = _share;
    _max[threadIdx.x] = row[threadIdx.x];
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        if (row[id] > _max[threadIdx.x]) {
          _max[threadIdx.x] = row[id];
        }
      }
    }

    int len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        if(_max[threadIdx.x + skip] > _max[threadIdx.x])
          _max[threadIdx.x] = _max[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }
    __syncthreads();
    float max = _max[0];
    __syncthreads();

    float* _sum = _share;// + blockDim.x;

    _sum[threadIdx.x] = 0.0f;
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        //row[id] = exp(row[id] - max);
        row[id] = __expf(row[id] - max);
        _sum[threadIdx.x] += row[id];
      }
    }

    len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        _sum[threadIdx.x] += _sum[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }

    __syncthreads();

    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        //row[id] = log(row[id]/_sum[0]);
        row[id] = __logf(row[id]/_sum[0]);
      }
    }
    __syncthreads();
    rowIdx += gridDim.x;
  }
}


Matrix& LogSoftmax(Matrix& Out) {
  int blocks = std::min(MAX_BLOCKS, (int)Out.dim(0));
  int threads = std::min(MAX_THREADS, (int)Out.dim(1));
  int shared = sizeof(float) * threads * 2;

  gLogSoftMax<<<blocks, 500, shared, CudaStreamHandler::GetStream()>>>
    (Out.data(), Out.dim(0), Out.dim(1));

  return Out;
}

__global__ void gSetColumn(float* d_in, int n_columns, int n_rows, int noColumn, float value) {
  int rowNumber = threadIdx.x  + blockDim.x * blockIdx.x;
  int index = noColumn + rowNumber * n_columns;

  if (index < n_columns * n_rows) {
    d_in[index] = value;
  }
}

void SetColumn(Matrix& In, int noColumn, float value) {
  int nColumns = In.dim(1);
  int nRows = In.dim(0);
  int nBlocks = nRows / 512 + ((nRows % 512 == 0) ?  0 : 1);
  int nThreads = std::min(512, nRows);

  gSetColumn<<<nBlocks, nThreads, 0, mblas::CudaStreamHandler::GetStream()>>>
    (In.data(), nColumns, nRows, noColumn, value);
}

__global__ void gFill(float* d_in, int size, float val) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < size) {
    d_in[index] = val;
  }
}

void Fill(Matrix& In, float value) {
  size_t size = In.size();
  int nThreads = std::min(512, (int)size);
  int nBlocks = (size / nThreads) + ((size % nThreads == 0) ? 0 : 1);

  gFill<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
    (In.data(), size, value);
}

__global__
void gMapMatrix(float* d_in, int numRows, int numCols, int mappingCols, const int* mapping, int i) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < numRows * numCols) {
    int batchIdx = tid / numCols;
    d_in[tid] *= mapping[mappingCols * batchIdx + i];
  }
}

void MapMatrix(Matrix& state, const DeviceVector<int>& mapping, size_t i)
{
  // blank out rows in the state matrix where the word position i does not exist
  // mapping is a concatenated array of 1 & 0 of each sentence in the batch to say whether word exists or not.

  int batchSize = state.dim(0);
  int stateLength = state.dim(1);
  int sentenceLength = mapping.size() / batchSize;

  int numThreads = std::min((int)state.size(), 512);
  int numBlocks = (state.size() / numThreads) + 1;

  float* d_in = state.data();
  const int* d_mapping = thrust::raw_pointer_cast(mapping.data());

  gMapMatrix<<<numBlocks, numThreads, 0, CudaStreamHandler::GetStream()>>>
    (d_in, batchSize, stateLength, sentenceLength, d_mapping, i);
}

__global__ void gLNormalization(float* out, const float* in, const float* alpha, const float* beta,
                                    int rows, int cols, float eps=0.00001) {
  extern __shared__ float _share[];

  for (int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if (j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0f;
      for (int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if (id < cols) {
          _sum[threadIdx.x] += sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if (threadIdx.x < (len >> 1)) {
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float mean = _sum[0] / cols;
      __syncthreads();

      float* _sqSum = _share + blockDim.x;

      _sqSum[threadIdx.x] = 0.0;
      for (int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = sp[id] - mean;
          so[id] = ex;
          _sqSum[threadIdx.x] += ex * ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sqSum[threadIdx.x] += _sqSum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float sigma = sqrtf(eps + (_sqSum[0] / cols));
      __syncthreads();

      for (int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if (beta != nullptr) {
            so[id] = alpha[id] * (so[id] / sigma) + beta[id];
          } else {
            so[id] = alpha[id] * (so[id] / sigma);
          }
        }
      }
    }
  }
}

void Normalization(Matrix& out, const Matrix& in, const Matrix& alpha, const Matrix& beta,
                       float eps) {
  int numThreads = std::min((int)in.dim(1), 512);

  out.Reshape(in.dim(0), in.dim(1), 1, 1);

  int rows = in.dim(0);
  int cols = in.dim(1);
  int numBlocks = std::min(rows, 65000);
  int shared = numThreads * sizeof(float) * 2;

  gLNormalization<<<numBlocks, numThreads, shared, CudaStreamHandler::GetStream()>>>
    (out.data(), in.data(), alpha.data(), beta.data(), rows, cols, eps);
}

void Normalization(Matrix& out, const Matrix& in, const Matrix& alpha, float eps) {
  int numThreads = std::min((int)in.dim(1), 512);

  out.Reshape(in.dim(0), in.dim(1), 1, 1);

  int rows = in.dim(0);
  int cols = in.dim(1);
  int numBlocks = std::min(rows, 65000);
  int shared = numThreads * sizeof(float) * 2;

  gLNormalization<<<numBlocks, numThreads, shared, CudaStreamHandler::GetStream()>>>
    (out.data(), in.data(), alpha.data(), nullptr, rows, cols, eps);
}

}  // namespace mblas
}  // namespace GPU
}  // namespace amunmt
