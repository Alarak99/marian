#include "hip/hip_runtime.h"
#include "gpu/mblas/matrix_functions.h"
#include "gpu/mblas/handles.h"

using namespace std;

namespace amunmt {
namespace GPU {
namespace mblas {

thread_local hipblasHandle_t* CublasHandler::handle_ = nullptr;
thread_local CudaStreamHandler* CudaStreamHandler::instance_ = nullptr;;

Matrix& Swap(Matrix& Out, Matrix& In) {
  Out.swap(In);
  return Out;
}

__global__ void gMean(MatrixWrapper<float> out,
                      const MatrixWrapper<float> in,
                      const MatrixWrapper<int>  mapping)
{
  assert(out.dim(0) == 1);
  // in = max sentence length, whatever, 1, batches
  // out = in, dim(0 = 1
  // mapping = max length * batches

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  //printf("id = %d in = %lu %lu %lu %lu = %lu %lu \n", id, in.dim(0), in.dim(1), in.dim(2), in.dim(3), in.size(), sizeof(in));

  if (id < out.size()) {
    size_t indices[SHAPE_SIZE];
    out.id2Indices(id, indices);
    assert(indices[0] == 0);
    //printf("%d -> %lu %lu %lu %lu \n", id, indices[0], indices[1], indices[2], indices[3]);

    size_t batch = indices[3];
    size_t startMapInd = batch * in.dim(0);

    float sum = 0.0f;
    int counter = 0;
    for (size_t row = 0; row < in.dim(0); ++row) {
      int isWord = mapping(row, batch, 0, 0);
      //printf("batch=%lu startMapInd=%lu  mapOffset=%lu -> %d \n", batch, startMapInd, mapOffset, isWord);
      if (isWord) {
        sum += in(row, indices[1], indices[2], indices[3]);
        ++counter;
      }
    }

    sum /= (float) counter;
    out[id] = sum;
  }
}

void Mean(Matrix& Out, const Matrix& In, const DeviceVector<int>& mapping) {
  size_t batchNum = Out.dim(0) * Out.dim(2) * Out.dim(3);
  size_t stateLength = Out.dim(1);
  size_t sentenceLength = (In.dim(0) * In.dim(2) * In.dim(3)) / batchNum;

  MatrixWrapper<float> outWrap(Out);
  MatrixWrapper<float> inWrap(In);

  MatrixWrapper<int> mappingWrap(mapping, sentenceLength, batchNum, 1, 1);

  size_t threads = MAX_THREADS;
  size_t blocks =  (outWrap.size() / threads) + ((outWrap.size() % threads == 0) ?  0 : 1);

  gMean<<<blocks, threads, 0, CudaStreamHandler::GetStream()>>>
    (outWrap, inWrap, mappingWrap);

}

__global__ void gWeightedMeanOld(float* d_out, const float* weights, const float* d_in, const int* mapping,
                              int numRows, int numCols, int srcLen) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < numRows * numCols) {
    int rowNo = id / numCols;
    int batchNo = mapping[rowNo];
    int statePos = id % numCols;

    float sum = 0.0f;
    for (int i = 0; i < srcLen; ++i) {
      sum += weights[rowNo * srcLen + i] * d_in[batchNo * srcLen * numCols + (i * numCols) + statePos];
    }

    d_out[id] = sum;
  }
}


__global__ void gWeightedMean(MatrixWrapper<float> out,
                              const MatrixWrapper<float> weights,
                              const MatrixWrapper<float> in,
                              const MatrixWrapper<int> mapping
                              )
{
  int numHypos = weights.dim(0);
  int states = in.dim(1);
  int srcLen = weights.dim(1);

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < numHypos * states) {
    int hypoInd = id / states;
    int batchInd = mapping[hypoInd];
    int stateInd = id % states;
    //printf("hypoInd=%d batchInd=%d stateInd=%d \n", hypoInd, batchInd, stateInd);

    float sum = 0.0f;
    for (uint i = 0; i < srcLen; ++i) {
      sum += weights(hypoInd, i, 0, 0) * in(i, stateInd, 0, batchInd);
    }

    out[id] = sum;
  }
}

void WeightedMean(Matrix& Out,const Matrix& Weights, const Matrix& In, const DeviceVector<int>& mapping) {
  int numHypos = Weights.dim(0);
  int states = In.dim(1);

  Out.Resize(numHypos, states);

  MatrixWrapper<float> outWrap(Out);
  MatrixWrapper<float> weightsWrap(Weights);
  MatrixWrapper<float> inWrap(In);
  MatrixWrapper<int> mappingWrap(mapping);

  int nThreads = MAX_THREADS;
  int nBlocks =  (Out.size() / MAX_THREADS) + ((Out.size() % MAX_THREADS == 0) ?  0 : 1);

  gWeightedMean<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
    (outWrap, weightsWrap, inWrap, mappingWrap);
  /*
  cerr << "nBlocks=" << nBlocks << endl;

  cerr << "Out=" << outWrap.Debug() << endl;
  cerr << "Weights=" << weightsWrap.Debug() << endl;
  cerr << "In=" << inWrap.Debug() << endl;
  cerr << "mapping=" << mapping.size() << endl;
  for (size_t i = 0; i < mapping.size(); ++i) {
    cerr << mapping[i] << " ";
  }
  cerr << endl << endl;
  */
}

Matrix& Transpose(Matrix& Out, const Matrix& In) {
  size_t m = In.dim(0);
  size_t n = In.dim(1);

  Out.Resize(n, m);

  float alpha = 1.0;
  float beta  = 0.0;

  hipblasSgeam(CublasHandler::GetHandle(), HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, In.data(), n,
              &beta, In.data(), n, Out.data(), m);

  return Out;
}

Matrix& Transpose(Matrix& Out) {
  Matrix Temp;
  Transpose(Temp, Out);
  Swap(Out, Temp);
  return Out;
}

Matrix& Concat(Matrix& Out, const Matrix& In) {
  size_t oldSize = Out.size();
  Out.Resize(Out.dim(0) + In.dim(0), Out.dim(1));

  mblas::copy(In.data(), In.size(), Out.data() + oldSize, hipMemcpyDeviceToDevice);

  return Out;
}

Matrix& Copy(Matrix& Out, const Matrix& In) {
  Out.Resize(In.dim(0), In.dim(1), In.dim(2), In.dim(3));

  mblas::copy(In.data(), In.size(), Out.data(), hipMemcpyDeviceToDevice);

  return Out;
}

__global__ void gPasteRows(  MatrixWrapper<float> outWrap,
                          const MatrixWrapper<float> inWrap,
                          int rowNo, int colNo)
{
  int inRows = inWrap.dim(0);
  int inCols = inWrap.dim(1);

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < inRows * inCols) {
    int outCols = outWrap.dim(1);

    int inRow = id / inCols;
    int inCol = id % inCols;

    //outWrap[outID] = inWrap[id];
    outWrap(rowNo, inCol + colNo, 0, inRow) = inWrap(inRow, inCol, 0, 0);
  }
}

void PasteRows(Matrix& Out, const Matrix& In, const size_t rowNo, size_t colNo)
{
  MatrixWrapper<float> outWrap(Out);
  MatrixWrapper<float> inWrap(In);

  int nThreads = MAX_THREADS;
  int nBlocks =  (In.size() / 512) + ((In.size() % 512 == 0) ?  0 : 1);

  gPasteRows<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
    (outWrap, inWrap, rowNo, colNo);

}

Matrix& PasteRow(Matrix& Out,
                 const Matrix& In,
                 const size_t r, const size_t c)
{
  size_t start = r * Out.dim(1) + c;

  mblas::copy(In.data(), In.size(), Out.data() + start, hipMemcpyDeviceToDevice);

  return Out;
}

Matrix& CopyRow(Matrix& Out,
                const Matrix& In,
                const size_t r, const size_t c) {
  size_t length = In.dim(1) - c;
  Out.Resize(1, length);
  size_t start = r * In.dim(1) + c;
  //size_t end   = start + length;

  //mblas::copy(In.begin() + start, In.begin() + end, Out.begin());
  mblas::copy(In.data() + start, length , Out.data(), hipMemcpyDeviceToDevice);

  return Out;
}

__global__ void gCopyRows(MatrixWrapper<float> outWrap,
                          const MatrixWrapper<float> inWrap,
                          const MatrixWrapper<size_t> indicesWrap,
                          float* out, const float* in)
{
  size_t numPairs = indicesWrap.size();
  size_t cols = inWrap.dim(1);

  size_t indicesInd = blockIdx.x;
  size_t inRow =indicesWrap[indicesInd];

  size_t colInd = threadIdx.x;
  while (colInd < outWrap.dim(1)) {
	  outWrap(indicesInd, colInd, 0, 0) = inWrap(inRow, colInd, 0, 0);
	  colInd += gridDim.x;
  }
}

Matrix& CopyRows(Matrix& Out,
                 const Matrix& In,
                 const DeviceVector<size_t>& indices)
{
  float* d_out = Out.data();
  const float* d_in = In.data();

  size_t numPairs = indices.size();

  MatrixWrapper<float> outWrap(Out);
  const MatrixWrapper<float> inWrap(In);
  const MatrixWrapper<size_t> indicesWrap(indices);

  int threads = std::min(MAX_THREADS, (int)In.dim(1));
  int blocks = std::min(MAX_BLOCKS, (int)numPairs);

  gCopyRows<<<blocks, threads, 0, CudaStreamHandler::GetStream()>>>
    (outWrap, inWrap, indicesWrap, d_out, d_in);

  return Out;
}


Matrix& Assemble(Matrix& Out,
                 const Matrix& In,
                 const DeviceVector<size_t>& indices) {
  Out.Resize(indices.size(), In.dim(1));
  //cerr << "Assemble=" << Out.Debug() << " " << In.Debug() << indices.size() << endl;

  CopyRows(Out, In, indices);
  return Out;
}

__global__ void gSlice(MatrixWrapper<float> outWrap,
						          const MatrixWrapper<float> inWrap,
                       size_t n, size_t dim)
{
  size_t row = blockIdx.x;

  size_t inCol = threadIdx.x + dim * n;
  size_t outCol = threadIdx.x;

  while (outCol < outWrap.dim(1)) {
    outWrap(row, outCol, 0, 0) = inWrap(row, inCol, 0, 0);

    inCol += gridDim.x;
    outCol += gridDim.x;
  }

}

Matrix& Slice(Matrix& Out,
              const Matrix& In,
              size_t n, size_t dim) {

  Out.Resize(In.dim(0), dim);

  MatrixWrapper<float> outWrap(Out);
  const MatrixWrapper<float> inWrap(In);

  int threads = std::min(MAX_THREADS, (int)dim);
  int blocks = std::min(MAX_BLOCKS, (int)In.dim(0));

  gSlice<<<blocks, threads, 0, CudaStreamHandler::GetStream()>>>
    (outWrap, inWrap, n, dim);

  return Out;
}

Matrix& Prod(hipblasHandle_t handle, Matrix& C, const Matrix& A, const Matrix& B,
             bool transA, bool transB) {
  Matrix::value_type alpha = 1.0;
  Matrix::value_type beta = 0.0;

  size_t m = A.dim(0);
  size_t k = A.dim(1);
  if(transA)
    std::swap(m, k);

  size_t l = B.dim(0);
  size_t n = B.dim(1);
  if(transB)
    std::swap(l, n);

  size_t lda = A.dim(1);
  size_t ldb = B.dim(1);
  size_t ldc = B.dim(1);

  if(transB)
    ldc = B.dim(0);

  C.Resize(m, n, A.dim(2), A.dim(3));
  //cerr << "C=" << C.Debug(1) << endl;

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  size_t m2 = A.dim(0) * A.dim(2) * A.dim(3);

  hipblasSgemm(handle, opB, opA,
              n, m2, k, &alpha, B.data(), ldb, A.data(), lda, &beta, C.data(), ldc);
  return C;
}

Matrix& Prod(Matrix& C, const Matrix& A, const Matrix& B,
             bool transA, bool transB) {

  //std::cerr << "1C=" << C.Debug() << std::endl;
  //std::cerr << "1A=" << A.Debug() << std::endl;
  //std::cerr << "1B=" << B.Debug() << std::endl;

  Matrix &ret = Prod(CublasHandler::GetHandle(), C, A, B, transA, transB);

  //std::cerr << "2C=" << C.Debug() << std::endl;
  return ret;
}

__global__ void gSoftMax(MatrixWrapper<float> outWrap,
                         const MatrixWrapper<int> batchIdsWrap,
                         const MatrixWrapper<int> srcMappingWrap)
{
  extern __shared__ float _share[];

  size_t numHypos = outWrap.dim(0);
  size_t srcLen = outWrap.dim(1);

  int hypoInd =  blockIdx.x;
  int origSrcPos = threadIdx.x;

  while (hypoInd < numHypos) {
    float* _max = _share;
    _max[origSrcPos] = outWrap(hypoInd, origSrcPos, 0, 0);
    for (int tid = 0; tid < srcLen; tid += blockDim.x) {
      int srcPos = tid + origSrcPos;
      if (srcPos < srcLen) {
        float value = outWrap(hypoInd, srcPos, 0, 0);

        int batch = batchIdsWrap[hypoInd];
        value *= srcMappingWrap(srcPos, batch, 0, 0);
        if (value > _max[origSrcPos]) {
          _max[origSrcPos] = value;
        }
      }
    }

    int len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (origSrcPos < (len >> 1)) {
        if(_max[origSrcPos + skip] > _max[origSrcPos])
          _max[origSrcPos] = _max[origSrcPos + skip];
      }
      len = (len + 1) >> 1;
    }
    __syncthreads();
    float max = _max[0];
    __syncthreads();

    float* _sum = _share;// + blockDim.x;
    _sum[origSrcPos] = 0.0f;
    for (int tid = 0; tid < srcLen; tid += blockDim.x) {
      int srcPos = tid + origSrcPos;
      if (srcPos < srcLen) {
        outWrap(hypoInd, srcPos, 0, 0) = __expf(outWrap(hypoInd, srcPos, 0, 0) - max);

        int batch = batchIdsWrap[hypoInd];
        outWrap(hypoInd, srcPos, 0, 0) *= srcMappingWrap(srcPos, batch, 0, 0);
        _sum[origSrcPos] += outWrap(hypoInd, srcPos, 0, 0);
      }
    }

    __syncthreads();

    len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (origSrcPos < (len >> 1)) {
        _sum[origSrcPos] += _sum[origSrcPos + skip];
      }
      len = (len + 1) >> 1;
    }

    __syncthreads();

    for (int tid = 0; tid < srcLen; tid += blockDim.x) {
      int srcPos = tid + origSrcPos;
      if (srcPos < srcLen) {
        outWrap(hypoInd, srcPos, 0, 0) /= _sum[0];
      }
    }
    __syncthreads();
    hypoInd += gridDim.x;
  }
}

Matrix& Softmax(Matrix& Out, const DeviceVector<int>& batchIds, const DeviceVector<int>& srcMapping, size_t batchSize)
{
  size_t srcSize = Out.dim(1);

  MatrixWrapper<float> outWrap(Out);
  const MatrixWrapper<int> batchIdsWrap(batchIds);
  const MatrixWrapper<int> srcMappingWrap(srcMapping, srcSize, batchSize, 1, 1);

  int blocks = batchSize;
  int threads = std::min(MAX_THREADS, (int)srcSize);
  int shared = sizeof(float) * threads;

  gSoftMax<<<blocks, threads, shared, CudaStreamHandler::GetStream()>>>
    (outWrap, batchIdsWrap, srcMappingWrap);

  return Out;
}

__global__ void gLogSoftMax(MatrixWrapper<float> outWrap)
{
  extern __shared__ float _share[];

  size_t rows = outWrap.dim(0);
  size_t cols = outWrap.dim(1);

  int rowIdx =  blockIdx.x;

  while (rowIdx < rows) {
    float* _max = _share;
    _max[threadIdx.x] = outWrap(rowIdx, threadIdx.x, 0, 0);
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        const float &val = outWrap(rowIdx, id, 0, 0);
        if (val > _max[threadIdx.x]) {
          _max[threadIdx.x] = val;
        }
      }
    }

    int len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        if(_max[threadIdx.x + skip] > _max[threadIdx.x])
          _max[threadIdx.x] = _max[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }
    __syncthreads();
    float max = _max[0];
    __syncthreads();

    float* _sum = _share;// + blockDim.x;

    _sum[threadIdx.x] = 0.0f;
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        //row[id] = exp(row[id] - max);
        float &val = outWrap(rowIdx, id, 0, 0);
        val = __expf(val - max);
        _sum[threadIdx.x] += val;
      }
    }

    len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        _sum[threadIdx.x] += _sum[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }

    __syncthreads();

    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        //row[id] = log(row[id]/_sum[0]);
        float &val = outWrap(rowIdx, id, 0, 0);
        val = __logf(val /_sum[0]);
      }
    }
    __syncthreads();
    rowIdx += gridDim.x;
  }
}


Matrix& LogSoftmax(Matrix& Out)
{
  MatrixWrapper<float> outWrap(Out);

  int blocks = std::min(MAX_BLOCKS, (int)Out.dim(0));
  int threads = std::min(MAX_THREADS, (int)Out.dim(1));
  int shared = sizeof(float) * threads;

  gLogSoftMax<<<blocks, threads, shared, CudaStreamHandler::GetStream()>>>
    (Out);

  return Out;
}

__global__ void gSetColumn(MatrixWrapper<float> inWrap, int noColumn, float value) {
  int n_rows = inWrap.dim(0);

  int rowNumber = threadIdx.x  + blockDim.x * blockIdx.x;

  if (rowNumber < n_rows) {
    inWrap(rowNumber, noColumn, 0, 0) = value;
  }
}

void SetColumn(Matrix& In, int noColumn, float value) {
  int nRows = In.dim(0);
  int nBlocks = nRows / MAX_THREADS + ((nRows % MAX_THREADS == 0) ?  0 : 1);
  int nThreads = std::min(MAX_THREADS, nRows);

  MatrixWrapper<float> inWrap(In);

  gSetColumn<<<nBlocks, nThreads, 0, mblas::CudaStreamHandler::GetStream()>>>
    (inWrap, noColumn, value);
}

__global__ void gFill(MatrixWrapper<float> inWrap, float val) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < inWrap.size()) {
    inWrap[index] = val;
  }
}

void Fill(Matrix& In, float value) {
  size_t size = In.size();

  if (value) {
    int nThreads = std::min(MAX_THREADS, (int)size);
    int nBlocks = (size / nThreads) + ((size % nThreads == 0) ? 0 : 1);

    MatrixWrapper<float> inWrap(In);

    gFill<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
      (inWrap, value);
  }
  else {
    HANDLE_ERROR(hipMemset(In.data(), 0, size * sizeof(float)));
  }

}

__global__
void gMapMatrix(MatrixWrapper<float> inWrap,
                const MatrixWrapper<int> mappingWrap,
                int mappingCols, int i)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < inWrap.size()) {
    int numCols = inWrap.dim(1);
    int batchIdx = tid / numCols;
    int col = tid % numCols;

    //inWrap[tid] *= mappingWrap(i, batchIdx, 0, 0);
    inWrap(batchIdx, col, 0, 0) *= mappingWrap(i, batchIdx, 0, 0); // [mappingCols * batchIdx + i];
  }
}

void MapMatrix(Matrix& state, const DeviceVector<int>& mapping, size_t i)
{
  // blank out rows in the state matrix where the word position i does not exist
  // mapping is a concatenated array of 1 & 0 of each sentence in the batch to say whether word exists or not.

  int batchSize = state.dim(0);
  int stateLength = state.dim(1);
  int sentenceLength = mapping.size() / batchSize;

  int numThreads = std::min((int)state.size(), MAX_THREADS);
  int numBlocks = (state.size() / numThreads) + 1;

  MatrixWrapper<float> stateWrap(state);
  MatrixWrapper<int> mappingWrap(mapping, sentenceLength, batchSize, 1, 1);

  gMapMatrix<<<numBlocks, numThreads, 0, CudaStreamHandler::GetStream()>>>
    (stateWrap, mappingWrap, sentenceLength, i);

  /*
  cerr << "nBlocks=" << numBlocks << endl;
  cerr << "nThreads=" << numThreads << endl;
  cerr << "stateWrap=" << stateWrap.Debug() << endl;
  cerr << "mapping=" << Debug(mapping, 2) << endl;
  cerr << "i=" << i << endl;
  cerr << std::endl;

  HANDLE_ERROR(hipDeviceSynchronize());
  */
}

__global__ void gLNormalization(MatrixWrapper<float> outWrap,
                                const MatrixWrapper<float> inWrap,
                                const MatrixWrapper<float> alphaWrap,
                                const MatrixWrapper<float> betaWrap,
                                float* out, const float* in, const float* alpha, const float* beta,
                                int rows, int cols, float eps=0.00001)
{
  extern __shared__ float _share[];

  for (int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if (j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0f;
      for (int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if (id < cols) {
          _sum[threadIdx.x] += sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if (threadIdx.x < (len >> 1)) {
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float mean = _sum[0] / cols;
      __syncthreads();

      float* _sqSum = _share + blockDim.x;

      _sqSum[threadIdx.x] = 0.0;
      for (int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = sp[id] - mean;
          so[id] = ex;
          _sqSum[threadIdx.x] += ex * ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sqSum[threadIdx.x] += _sqSum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float sigma = sqrtf(eps + (_sqSum[0] / cols));
      __syncthreads();

      for (int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if (beta != nullptr) {
            so[id] = alpha[id] * (so[id] / sigma) + beta[id];
          } else {
            so[id] = alpha[id] * (so[id] / sigma);
          }
        }
      }
    }
  }
}

void Normalization(Matrix& out, const Matrix& in, const Matrix& alpha, const Matrix& beta,
                       float eps) {
  int numThreads = std::min((int)in.dim(1), 512);

  out.Reshape(in.dim(0), in.dim(1), 1, 1);

  int rows = in.dim(0);
  int cols = in.dim(1);
  int numBlocks = std::min(rows, 65000);
  int shared = numThreads * sizeof(float) * 2;

  MatrixWrapper<float> outWrap(out);
  const MatrixWrapper<float> inWrap(in);
  const MatrixWrapper<float> alphaWrap(alpha);
  const MatrixWrapper<float> betaWrap(beta);

  gLNormalization<<<numBlocks, numThreads, shared, CudaStreamHandler::GetStream()>>>
    (outWrap, inWrap, alphaWrap, betaWrap, out.data(), in.data(), alpha.data(), beta.data(), rows, cols, eps);
}

void Normalization(Matrix& out, const Matrix& in, const Matrix& alpha, float eps)
{
  int numThreads = std::min((int)in.dim(1), 512);

  out.Reshape(in.dim(0), in.dim(1), 1, 1);

  int rows = in.dim(0);
  int cols = in.dim(1);
  int numBlocks = std::min(rows, 65000);
  int shared = numThreads * sizeof(float) * 2;

  MatrixWrapper<float> outWrap(out);
  const MatrixWrapper<float> inWrap(in);
  const MatrixWrapper<float> alphaWrap(alpha);
  const MatrixWrapper<float> betaWrap;

  gLNormalization<<<numBlocks, numThreads, shared, CudaStreamHandler::GetStream()>>>
    (outWrap, inWrap, alphaWrap, betaWrap, out.data(), in.data(), alpha.data(), nullptr, rows, cols, eps);
}

}  // namespace mblas
}  // namespace GPU
}  // namespace amunmt
