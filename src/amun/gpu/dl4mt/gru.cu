#include "hip/hip_runtime.h"
#include "gru.h"

namespace amunmt {
namespace GPU {

__global__ void gElementwiseOps(mblas::MatrixWrapper<float> outWrap,
                                const mblas::MatrixWrapper<float> stateWrap,
                                const mblas::MatrixWrapper<float> ruhWrap,
                                const mblas::MatrixWrapper<float> tempWrap,
                                const mblas::MatrixWrapper<float> bWrap,
                                const mblas::MatrixWrapper<float> bx1Wrap,
                                const mblas::MatrixWrapper<float> bx2Wrap)
{
  const uint rows = stateWrap.dim(0) * stateWrap.dim(2) * stateWrap.dim(3);
  const uint cols = stateWrap.dim(1);
  assert(blockIdx.x < rows);
  assert(ruhWrap.dim(1) == cols * 3);

  const float* rowRuh = &ruhWrap[0] + blockIdx.x * cols * 3;
  const float* rowT = &tempWrap[0] + blockIdx.x * cols * 3;

  const float* rowH = rowRuh + 2 * cols;
  const float* rowT2 = rowT + 2 * cols;

  for(int tid = 0; tid < cols; tid += blockDim.x) {
    int i = tid + threadIdx.x;
    if(i < cols) {
      float ev1 = expf(-(rowRuh[i]
                         + bWrap[i]
                         + tempWrap[blockIdx.x * tempWrap.dim(1) + i]
                        )
                      );
      float r = 1.0f / (1.0f + ev1);

      int k = i + cols;
      float ev2 = expf(-(rowRuh[k]
                         + bWrap[k]
                         + tempWrap[blockIdx.x * tempWrap.dim(1) + k]
                        )
                      );
      float u = 1.0f / (1.0f + ev2);

      float hv = rowH[i] + bx1Wrap[i];
      float t2v = rowT2[i] + bx2Wrap[i];
      hv = tanhf(hv + r * t2v);
      outWrap[blockIdx.x * cols + i] = (1.0f - u) * hv + u * stateWrap[blockIdx.x * cols + i];
    }
  }
}

}
}

