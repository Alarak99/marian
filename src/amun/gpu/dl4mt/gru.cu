#include "hip/hip_runtime.h"
#include "gru.h"

namespace amunmt {
namespace GPU {

__global__ void gElementwiseOps(mblas::MatrixWrapper<float> outWrap,
                                const mblas::MatrixWrapper<float> stateWrap,
                                const mblas::MatrixWrapper<float> ruhWrap,
                                const mblas::MatrixWrapper<float> tempWrap,
                                const mblas::MatrixWrapper<float> bWrap,
                                const mblas::MatrixWrapper<float> bx1Wrap,
                                const mblas::MatrixWrapper<float> bx2Wrap)
{
  const uint rows = stateWrap.dim(0) * stateWrap.dim(2) * stateWrap.dim(3);
  const uint cols = stateWrap.dim(1);
  assert(blockIdx.x < rows);
  assert(ruhWrap.dim(1) == cols * 3);

  for(int tid = 0; tid < cols; tid += blockDim.x) {
    int i = tid + threadIdx.x;
    if(i < cols) {
      float ev1 = expf(-(ruhWrap[blockIdx.x * ruhWrap.dim(1) + i]
                         + bWrap[i]
                         + tempWrap[blockIdx.x * tempWrap.dim(1) + i]
                        )
                      );
      float r = 1.0f / (1.0f + ev1);

      int k = i + cols;
      float ev2 = expf(-(ruhWrap[blockIdx.x * ruhWrap.dim(1) + k]
                         + bWrap[k]
                         + tempWrap[blockIdx.x * tempWrap.dim(1) + k]
                        )
                      );
      float u = 1.0f / (1.0f + ev2);

      //float hv = rowH[i] + bx1Wrap[i];
      float hv = ruhWrap[blockIdx.x * ruhWrap.dim(1) + 2*cols + i]
               + bx1Wrap[i];

      //float t2v = rowT2[i] + bx2Wrap[i];
      float t2v = tempWrap[blockIdx.x * tempWrap.dim(1) + 2*cols + i]
                + bx2Wrap[i];

      hv = tanhf(hv + r * t2v);
      outWrap[blockIdx.x * cols + i] = (1.0f - u) * hv + u * stateWrap[blockIdx.x * cols + i];
    }
  }
}

}
}

