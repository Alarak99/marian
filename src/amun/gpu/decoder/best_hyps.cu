#include "best_hyps.h"
#include "common/histories.h"
#include "common/hypothesis.h"

using namespace std;

namespace amunmt {
namespace GPU {

BestHyps::BestHyps(const God &god)
      : BestHypsBase(god),
        keys_(god.Get<size_t>("beam-size") * god.Get<size_t>("mini-batch")),
        costs_(god.Get<size_t>("beam-size") * god.Get<size_t>("mini-batch")),
        maxBeamSize_(god.Get<uint>("beam-size"))
{
  if (!god_.UseFusedSoftmax()) {
    NthElement *obj = new NthElement(god.Get<size_t>("beam-size"), god.Get<size_t>("mini-batch"));
    nthElement_.reset(obj);
  }
}

BestHyps::~BestHyps()
{
  //cerr << "~BestHyps" << endl;
}

void BestHyps::DisAllowUNK(mblas::Matrix& Prob) {
  SetColumn(Prob, UNK_ID, std::numeric_limits<float>::lowest());
}

void BestHyps::FindBests(const Histories& histories,
                          mblas::Matrix& Probs,
                          std::vector<float>& outCosts,
                          std::vector<unsigned>& outKeys)
{
  nthElement_->getNBestList(histories, Probs, outCosts, outKeys);
}

// fast fused softmax and nth_element
void BestHyps::FindBests(const Histories& histories,
                        mblas::Matrix& Probs,
                        mblas::Vector<NthOutBatch> &nBest,
                        std::vector<float>& outCosts,
                        std::vector<unsigned>& outKeys)
{
  getNBestList(histories, Probs, nBest, outCosts, outKeys);
}

std::vector<SoftAlignmentPtr> BestHyps::GetAlignments(const std::vector<ScorerPtr>& scorers,
                                            size_t hypIndex)
{
  std::vector<SoftAlignmentPtr> alignments;
  for (auto& scorer : scorers) {
    if (GPU::EncoderDecoder* encdec = dynamic_cast<GPU::EncoderDecoder*>(scorer.get())) {
      const mblas::Matrix &attention = encdec->GetAttention();
      size_t attLength = attention.dim(1);

      SoftAlignment *softAlignment = new SoftAlignment(attLength);
      mblas::copy(
          attention.data() + hypIndex * attLength,
          attLength,
          softAlignment->data(),
          hipMemcpyDeviceToHost
      );

      alignments.emplace_back(softAlignment);
    } else {
      amunmt_UTIL_THROW2("Return Alignment is allowed only with Nematus scorer.");
    }
  }
  return alignments;
}

//////////////////////////////////////////////////////////////////////////
void BestHyps::getNBestList(const Histories& histories,
                  mblas::Matrix& Probs,
                  mblas::Vector<NthOutBatch> &nBest,
                  std::vector<float>& outCosts,
                  std::vector<uint>& outKeys) const
{
  GetPairs(nBest, outKeys, outCosts);
  assert(outCosts.size() == outKeys.size());

  /*
  cerr << "outCosts/outKeys=";
  for (size_t i = 0; i < outKeys.size(); ++i) {
    cerr << "(" << outCosts[i] << "," << outKeys[i] << ") ";
  }
  cerr << endl;
  */
  //cerr << endl;
}

void BestHyps::GetPairs(mblas::Vector<NthOutBatch> &nBest,
              std::vector<uint>& outKeys,
              std::vector<float>& outValues) const
{
  //cerr << "top=" << top2.size() << " nBest=" << nBest.size() << endl;
  outKeys.resize(nBest.size());
  outValues.resize(nBest.size());

  std::vector<NthOutBatch> hostVec(nBest.size());
  mblas::copy(nBest.data(), nBest.size(), hostVec.data(), hipMemcpyDeviceToHost);

  for (size_t i = 0; i < nBest.size(); ++i) {
    outKeys[i] = hostVec[i].ind;
    outValues[i] = hostVec[i].score;
  }
}

/////////////////////////////////////////////////////////////////////////////////////
// const-batch2
std::vector<SoftAlignmentPtr> BestHyps::GetAlignments(Scorer &scorer,
                                            size_t hypIndex)
{
  std::vector<SoftAlignmentPtr> alignments;
  GPU::EncoderDecoder &encdec = static_cast<GPU::EncoderDecoder&>(scorer);
  const mblas::Matrix &attention = encdec.GetAttention();
  size_t attLength = attention.dim(1);

  SoftAlignment *softAlignment = new SoftAlignment(attLength);
  mblas::copy(
      attention.data() + hypIndex * attLength,
      attLength,
      softAlignment->data(),
      hipMemcpyDeviceToHost
  );

  alignments.emplace_back(softAlignment);

  return alignments;

}

// standard nth_element
void  BestHyps::CalcBeam(
    const Hypotheses& prevHyps,
    Scorer &scorer,
    const Words& filterIndices,
    Histories& histories)
{
  BEGIN_TIMER("CalcBeam");
  using namespace mblas;

  mblas::Matrix& Probs = static_cast<mblas::Matrix&>(scorer.GetProbs());

  std::vector<float> vCosts;
  for (const HypothesisPtr &h : prevHyps) {
    if (h) {
      vCosts.push_back(h->GetCost());
    }
  }

  mblas::copy(vCosts.data(),
              vCosts.size(),
              costs_.data(),
              hipMemcpyHostToDevice);
  //mblas::copy(vCosts.begin(), vCosts.end(), costs_.begin());

  size_t beamSizeSum = histories.Sum();;

  std::vector<float> bestCosts;
  std::vector<unsigned> bestKeys;

  if (god_.UseFusedSoftmax()) {
    const mblas::Matrix& b4 = *static_cast<const mblas::Matrix*>(scorer.GetBias());
    mblas::Vector<NthOutBatch> &nBest = *static_cast<mblas::Vector<NthOutBatch>*>(scorer.GetNBest());
    nBest.newSize(beamSizeSum);

    BEGIN_TIMER("GetProbs.LogSoftmaxAndNBest");
    mblas::LogSoftmaxAndNBest(nBest, Probs, b4, costs_, forbidUNK_, maxBeamSize_, histories, beamSizeSum);
    PAUSE_TIMER("GetProbs.LogSoftmaxAndNBest");
    //std::cerr << "2Probs=" << Probs.Debug(1) << std::endl;

    FindBests(histories, Probs, nBest, bestCosts, bestKeys);
  }
  else {
    BroadcastVecColumn(weights_.at(scorer.GetName()) * _1 + _2, Probs, costs_);

    if (forbidUNK_) {
      DisAllowUNK(Probs);
    }

    FindBests(histories, Probs, bestCosts, bestKeys);
  }

  std::vector<std::vector<float>> breakDowns;
  if (god_.ReturnNBestList()) {
      breakDowns.push_back(bestCosts);
  }

  std::map<size_t, size_t> batchMap;
  size_t tmp = 0;
  for (size_t batchID = 0; batchID < histories.size(); ++batchID) {
    for (size_t t = 0; t < histories.GetBeamSize(batchID); ++t) {
      batchMap[tmp++] = batchID;
    }
  }

  for (size_t i = 0; i < beamSizeSum; i++) {
    size_t wordIndex = bestKeys[i] % Probs.dim(1);
    if (isInputFiltered_) {
      wordIndex = filterIndices[wordIndex];
    }

    size_t hypIndex  = bestKeys[i] / Probs.dim(1);
    float cost = bestCosts[i];

    HypothesisPtr hyp;
    if (returnAttentionWeights_) {
      hyp.reset(new Hypothesis(prevHyps[hypIndex], wordIndex, hypIndex, cost,
                               GetAlignments(scorer, hypIndex)));
    } else {
      hyp.reset(new Hypothesis(prevHyps[hypIndex], wordIndex, hypIndex, cost));
    }

    //cerr << "god_.ReturnNBestList()=" << god_.ReturnNBestList() << endl;
    if(god_.ReturnNBestList()) {
      hyp->GetCostBreakdown().resize(1);
      float sum = 0;
      hyp->GetCostBreakdown()[0] = breakDowns[0][i];
      hyp->GetCostBreakdown()[0] -= sum;
      hyp->GetCostBreakdown()[0] /= weights_.at(scorer.GetName());
    }

    size_t batchInd = batchMap[i];
    //cerr << "batchInd=" << batchInd << endl;
    HistoriesElementPtr &ele = histories.Get(batchInd);
    assert(ele);
    ele->GetHypotheses().push_back(hyp);
  }

  PAUSE_TIMER("CalcBeam");
}

} // namespace
}
