#include "best_hyps.h"
#include "common/histories.h"
#include "common/hypothesis.h"

using namespace std;

namespace amunmt {
namespace GPU {

BestHyps::BestHyps(const God &god)
      : BestHypsBase(god),
        keys_(god.Get<size_t>("beam-size") * god.Get<size_t>("mini-batch")),
        costs_(god.Get<size_t>("beam-size") * god.Get<size_t>("mini-batch")),
        maxBeamSize_(god.Get<uint>("beam-size"))
{
  if (!god_.UseFusedSoftmax()) {
    NthElement *obj = new NthElement(god.Get<size_t>("beam-size"), god.Get<size_t>("mini-batch"));
    nthElement_.reset(obj);
  }
}

BestHyps::~BestHyps()
{
  //cerr << "~BestHyps" << endl;
}

void BestHyps::DisAllowUNK(mblas::Matrix& Prob) {
  SetColumn(Prob, UNK_ID, std::numeric_limits<float>::lowest());
}

void BestHyps::FindBests(const Histories& beamSizes,
                          mblas::Matrix& Probs,
                          std::vector<float>& outCosts,
                          std::vector<unsigned>& outKeys)
{
  nthElement_->getNBestList(beamSizes, Probs, outCosts, outKeys);
}

// fast fused softmax and nth_element
void BestHyps::FindBests(const Histories& beamSizes,
                        mblas::Matrix& Probs,
                        mblas::Vector<NthOutBatch> &nBest,
                        std::vector<float>& outCosts,
                        std::vector<unsigned>& outKeys)
{
  getNBestList(beamSizes, Probs, nBest, outCosts, outKeys);
}

std::vector<SoftAlignmentPtr> BestHyps::GetAlignments(const std::vector<ScorerPtr>& scorers,
                                            size_t hypIndex)
{
  std::vector<SoftAlignmentPtr> alignments;
  for (auto& scorer : scorers) {
    if (GPU::EncoderDecoder* encdec = dynamic_cast<GPU::EncoderDecoder*>(scorer.get())) {
      const mblas::Matrix &attention = encdec->GetAttention();
      size_t attLength = attention.dim(1);

      SoftAlignment *softAlignment = new SoftAlignment(attLength);
      mblas::copy(
          attention.data() + hypIndex * attLength,
          attLength,
          softAlignment->data(),
          hipMemcpyDeviceToHost
      );

      alignments.emplace_back(softAlignment);
    } else {
      amunmt_UTIL_THROW2("Return Alignment is allowed only with Nematus scorer.");
    }
  }
  return alignments;
}

//////////////////////////////////////////////////////////////////////////
void BestHyps::getNBestList(const Histories& beamSizes,
                  mblas::Matrix& Probs,
                  mblas::Vector<NthOutBatch> &nBest,
                  std::vector<float>& outCosts,
                  std::vector<uint>& outKeys) const
{
  GetPairs(nBest, outKeys, outCosts);
  assert(outCosts.size() == outKeys.size());

  /*
  cerr << "outCosts/outKeys=";
  for (size_t i = 0; i < outKeys.size(); ++i) {
    cerr << "(" << outCosts[i] << "," << outKeys[i] << ") ";
  }
  cerr << endl;
  */
  //cerr << endl;
}

void BestHyps::GetPairs(mblas::Vector<NthOutBatch> &nBest,
              std::vector<uint>& outKeys,
              std::vector<float>& outValues) const
{
  //cerr << "top=" << top2.size() << " nBest=" << nBest.size() << endl;
  outKeys.resize(nBest.size());
  outValues.resize(nBest.size());

  std::vector<NthOutBatch> hostVec(nBest.size());
  mblas::copy(nBest.data(), nBest.size(), hostVec.data(), hipMemcpyDeviceToHost);

  for (size_t i = 0; i < nBest.size(); ++i) {
    outKeys[i] = hostVec[i].ind;
    outValues[i] = hostVec[i].score;
  }
}

/////////////////////////////////////////////////////////////////////////////////////
// const-batch2
std::vector<SoftAlignmentPtr> BestHyps::GetAlignments(Scorer &scorer,
                                            size_t hypIndex)
{
  std::vector<SoftAlignmentPtr> alignments;
  GPU::EncoderDecoder &encdec = static_cast<GPU::EncoderDecoder&>(scorer);
  const mblas::Matrix &attention = encdec.GetAttention();
  size_t attLength = attention.dim(1);

  SoftAlignment *softAlignment = new SoftAlignment(attLength);
  mblas::copy(
      attention.data() + hypIndex * attLength,
      attLength,
      softAlignment->data(),
      hipMemcpyDeviceToHost
  );

  alignments.emplace_back(softAlignment);

  return alignments;

}

// standard nth_element
void  BestHyps::CalcBeam(
    const Hypotheses& prevHyps,
    Scorer &scorer,
    const Words& filterIndices,
    HypothesesBatch& beams,
    Histories& beamSizes)
{
  BEGIN_TIMER("CalcBeam");
  using namespace mblas;
  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
  cerr << "BestHyps::CalcBeam1" << endl;

  mblas::Matrix& Probs = static_cast<mblas::Matrix&>(scorer.GetProbs());

  std::vector<float> vCosts;
  for (const HypothesisPtr &h : prevHyps) {
    if (h) {
      vCosts.push_back(h->GetCost());
    }
  }
  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
  cerr << "BestHyps::CalcBeam2" << endl;

  mblas::copy(vCosts.data(),
              vCosts.size(),
              costs_.data(),
              hipMemcpyHostToDevice);
  //mblas::copy(vCosts.begin(), vCosts.end(), costs_.begin());
  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
  cerr << "BestHyps::CalcBeam3" << endl;

  size_t beamSizeSum = beamSizes.Sum();;

  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
  cerr << "BestHyps::CalcBeam4" << endl;

  std::vector<float> bestCosts;
  std::vector<unsigned> bestKeys;

  if (god_.UseFusedSoftmax()) {
    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam5" << endl;
    const mblas::Matrix& b4 = *static_cast<const mblas::Matrix*>(scorer.GetBias());
    mblas::Vector<NthOutBatch> &nBest = *static_cast<mblas::Vector<NthOutBatch>*>(scorer.GetNBest());
    nBest.newSize(beamSizeSum);

    BEGIN_TIMER("GetProbs.LogSoftmaxAndNBest");
    mblas::LogSoftmaxAndNBest(nBest, Probs, b4, costs_, forbidUNK_, maxBeamSize_, beamSizes, beamSizeSum);
    PAUSE_TIMER("GetProbs.LogSoftmaxAndNBest");
    //std::cerr << "2Probs=" << Probs.Debug(1) << std::endl;

    FindBests(beamSizes, Probs, nBest, bestCosts, bestKeys);
    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam6" << endl;
  }
  else {
    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam7" << endl;
    BroadcastVecColumn(weights_.at(scorer.GetName()) * _1 + _2, Probs, costs_);

    if (forbidUNK_) {
      DisAllowUNK(Probs);
    }

    FindBests(beamSizes, Probs, bestCosts, bestKeys);
    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam8" << endl;
  }

  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
  cerr << "BestHyps::CalcBeam9" << endl;

  std::vector<std::vector<float>> breakDowns;
  if (god_.ReturnNBestList()) {
      breakDowns.push_back(bestCosts);
  }

  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
  cerr << "BestHyps::CalcBeam10" << endl;

  std::map<size_t, size_t> batchMap;
  size_t tmp = 0;
  for (size_t batchID = 0; batchID < beamSizes.size(); ++batchID) {
    for (size_t t = 0; t < beamSizes.GetBeamSize(batchID); ++t) {
      batchMap[tmp++] = batchID;
    }
  }

  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
  cerr << "BestHyps::CalcBeam11" << endl;

  for (size_t i = 0; i < beamSizeSum; i++) {
    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam12" << endl;

    size_t wordIndex = bestKeys[i] % Probs.dim(1);
    if (isInputFiltered_) {
      wordIndex = filterIndices[wordIndex];
    }

    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam12.1" << endl;

    size_t hypIndex  = bestKeys[i] / Probs.dim(1);
    float cost = bestCosts[i];

    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam12.2" << endl;
    cerr << "prevHyps=" << prevHyps.size() << endl;
    cerr << "hypIndex=" << hypIndex << endl;

    HypothesisPtr hyp;
    if (returnAttentionWeights_) {
      hyp.reset(new Hypothesis(prevHyps[hypIndex], wordIndex, hypIndex, cost,
                               GetAlignments(scorer, hypIndex)));
    } else {
      hyp.reset(new Hypothesis(prevHyps[hypIndex], wordIndex, hypIndex, cost));
    }

    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam12.3" << endl;

    //cerr << "god_.ReturnNBestList()=" << god_.ReturnNBestList() << endl;
    if(god_.ReturnNBestList()) {
      hyp->GetCostBreakdown().resize(1);
      float sum = 0;
      hyp->GetCostBreakdown()[0] = breakDowns[0][i];
      hyp->GetCostBreakdown()[0] -= sum;
      hyp->GetCostBreakdown()[0] /= weights_.at(scorer.GetName());
    }

    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam12.4" << endl;

    beams[batchMap[i]].push_back(hyp);

    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam12.5" << endl;

    HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
    cerr << "BestHyps::CalcBeam13" << endl;
  }

  HANDLE_ERROR( hipStreamSynchronize(mblas::CudaStreamHandler::GetStream()));
  cerr << "BestHyps::CalcBeam14" << endl;

  PAUSE_TIMER("CalcBeam");
}

} // namespace
}
