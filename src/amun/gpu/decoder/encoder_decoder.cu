// -*- mode: c++; tab-width: 2; indent-tabs-mode: nil -*-
#include <iostream>

#include "common/god.h"
#include "common/sentences.h"

#include "encoder_decoder.h"
#include "gpu/mblas/matrix_functions.h"
#include "gpu/dl4mt/dl4mt.h"
#include "gpu/decoder/encoder_decoder_state.h"
#include "gpu/decoder/best_hyps.h"

using namespace std;

namespace amunmt {
namespace GPU {

EncoderDecoder::EncoderDecoder(
		const God &god,
		const std::string& name,
        const YAML::Node& config,
        size_t tab,
        const Weights& model)
  : Scorer(name, config, tab),
    model_(model),
    encoder_(new Encoder(model_)),
    decoder_(new Decoder(god, model_)),
    indices_(god.Get<size_t>("beam-size"))
{}

State* EncoderDecoder::NewState() const {
  return new EDState();
}

void EncoderDecoder::SetSource(const Sentences& source) {
  encoder_->GetContext(source, tab_, sourceContext_, batchMapping_);
  //cerr << "GPU sourceContext_=" << sourceContext_.Debug(1) << endl;
}

void EncoderDecoder::BeginSentenceState(State& state, size_t batchSize) {
  EDState& edState = state.get<EDState>();
  decoder_->EmptyState(edState.GetStates(), sourceContext_, batchSize, batchMapping_);

  decoder_->EmptyEmbedding(edState.GetEmbeddings(), batchSize);
}

void EncoderDecoder::Decode(const God &god, const State& in, State& out, const std::vector<uint>& beamSizes) {
  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();

  decoder_->Decode(edOut.GetStates(),
                     edIn.GetStates(),
                     edIn.GetEmbeddings(),
                     sourceContext_,
                     batchMapping_,
                     beamSizes);
}

void EncoderDecoder::AssembleBeamState(const State& in,
                               const Beam& beam,
                               State& out) {
  std::vector<size_t> beamWords;
  std::vector<size_t> beamStateIds;
  for (const HypothesisPtr &h : beam) {
     beamWords.push_back(h->GetWord());
     beamStateIds.push_back(h->GetPrevStateIndex());
  }
  //cerr << "beamWords=" << Debug(beamWords, 2) << endl;
  //cerr << "beamStateIds=" << Debug(beamStateIds, 2) << endl;

  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();
  indices_.resize(beamStateIds.size());
  thrust::host_vector<size_t> tmp = beamStateIds;

  mblas::copy(thrust::raw_pointer_cast(tmp.data()),
      beamStateIds.size(),
      thrust::raw_pointer_cast(indices_.data()),
      hipMemcpyHostToDevice);
  //cerr << "indices_=" << mblas::Debug(indices_, 2) << endl;

  mblas::Assemble(edOut.GetStates(), edIn.GetStates(), indices_);
  //cerr << "edOut.GetStates()=" << edOut.GetStates().Debug(1) << endl;

  //cerr << "beamWords=" << Debug(beamWords, 2) << endl;
  decoder_->Lookup(edOut.GetEmbeddings(), beamWords);
  //cerr << "edOut.GetEmbeddings()=" << edOut.GetEmbeddings().Debug(1) << endl;
}

void EncoderDecoder::GetAttention(mblas::Matrix& Attention) {
  decoder_->GetAttention(Attention);
}

BaseMatrix& EncoderDecoder::GetProbs() {
  return decoder_->GetProbs();
}

mblas::Matrix& EncoderDecoder::GetAttention() {
  return decoder_->GetAttention();
}

size_t EncoderDecoder::GetVocabSize() const {
  return decoder_->GetVocabSize();
}

void EncoderDecoder::Filter(const std::vector<size_t>& filterIds) {
  decoder_->Filter(filterIds);
}

EncoderDecoder::~EncoderDecoder() {}

////////////////////////////////////////////
EncoderDecoderLoader::EncoderDecoderLoader(const std::string name,
                     const YAML::Node& config)
 : Loader(name, config) {}

void EncoderDecoderLoader::Load(const God &god) {
  std::string path = Get<std::string>("path");
  std::vector<size_t> devices = god.Get<std::vector<size_t>>("devices");

  size_t maxDeviceId = 0;
  for (size_t i = 0; i < devices.size(); ++i) {
    if (devices[i] > maxDeviceId) {
      maxDeviceId = devices[i];
    }
  }

  ThreadPool devicePool(devices.size());
  weights_.resize(maxDeviceId + 1);

  for(auto d : devices) {
    devicePool.enqueue([d, &path, this] {
        LOG(info, "Loading model {} onto gpu {}", path, d);
        HANDLE_ERROR(hipSetDevice(d));
        weights_[d].reset(new Weights(path, d));
      });
  }
}

EncoderDecoderLoader::~EncoderDecoderLoader()
{
  for (size_t d = 0; d < weights_.size(); ++d) {
    const Weights *weights = weights_[d].get();
    if (weights) {
      HANDLE_ERROR(hipSetDevice(d));
      weights_[d].reset(nullptr);
    }
  }
}

ScorerPtr EncoderDecoderLoader::NewScorer(const God &god, const DeviceInfo &deviceInfo) const {
  size_t d = deviceInfo.deviceId;

  HANDLE_ERROR(hipSetDevice(d));
  size_t tab = Has("tab") ? Get<size_t>("tab") : 0;
  return ScorerPtr(new EncoderDecoder(god, name_, config_,
                                      tab, *weights_[d]));
}

BestHypsBasePtr EncoderDecoderLoader::GetBestHyps(const God &god) const {
  return BestHypsBasePtr(new GPU::BestHyps(god));
}

}
}

