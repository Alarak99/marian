
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
//#include <cuda.h>
#include <stdio.h>
#include <chrono>
#include <hiprand/hiprand.h>
#include <hipblas.h>

using namespace std;


///////////////////////////////////////////////////////////////////////////////

hipStream_t stream;
hipblasHandle_t handle;

///////////////////////////////////////////////////////////////////////////////

void HandleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    std::cerr << "ERROR: " << hipGetErrorString(err) << " in " << file << " at line " << line << std::endl;
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

///////////////////////////////////////////////////////////////////////////////
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

     // Fill the array with random numbers on the device
     /* curandGenerateUniform(prng, A, nr_rows_A * nr_cols_A); */
}

///////////////////////////////////////////////////////////////////////////////

void gpu_blas_mmul(const float *A,
                   const float *B,
                   float *C,
                   const int m,
                   const int k,
                   const int n)
{
  int lda=m,ldb=k,ldc=m;

  float alpha = 1.0;
  float beta = 0.0;

  // Do the actual multiplication
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
      A, lda, B, ldb, &beta, C, ldc);

}

///////////////////////////////////////////////////////////////////////////////
void testBatchMultiply(int batchSize, int numIter, hipblasMath_t mathMode) 
{
  hipblasStatus_t stat = hipblasSetMathMode(handle, mathMode);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("hipblasSetMathMode failed\n");
    abort();
  }

  // Allocate 3 arrays on CPU
  int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

  nr_rows_A = batchSize;
  nr_cols_A = 512;
  nr_rows_B = 512;
  nr_cols_B = 85000;
  nr_rows_C = batchSize;
  nr_cols_C = 85000;

  // Allocate 3 arrays on GPU
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
  hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
  hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

  // Fill the arrays A and B on GPU with random numbers
  GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
  GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

  hipStreamSynchronize(stream);

  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();

  for (size_t i = 0; i < numIter; ++i) {
    // Multiply A and B on GPU
    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
  }

  hipStreamSynchronize(stream);
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  std::cout << "batchSize=" << batchSize 
            << " mathMode=" << mathMode
            << " time: " << elapsed_seconds.count() << endl;
 
  //Free GPU memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);  

}

///////////////////////////////////////////////////////////////////////////////

int main()
{
  cerr << "Starting" << endl;

  HANDLE_ERROR( hipStreamCreate(&stream));

  hipblasStatus_t stat;
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("hipblasCreate initialization failed\n");
    abort();
  }

  stat = hipblasSetStream(handle, stream);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("hipblasSetStream initialization failed\n");
    abort();
  }

  for (int batchSize = 640; batchSize > 0; --batchSize) {
    testBatchMultiply(batchSize, 10000, HIPBLAS_DEFAULT_MATH);
    testBatchMultiply(batchSize, 10000, HIPBLAS_TENSOR_OP_MATH);
  }

  hipblasDestroy(handle);
  HANDLE_ERROR(hipStreamDestroy(stream));

  cerr << "Finished" << endl;
}
